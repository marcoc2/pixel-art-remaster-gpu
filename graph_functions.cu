
#include <hip/hip_runtime.h>
//#include <cuda.h>

/** Check bit value */
#define CHECK_BIT( var, pos ) ( ( var ) & ( 1 << ( pos ) ) )

/** Clean bit then set */
#define SET_BIT( var, pos, data ) var &= ~( 1 << pos );var |= data << pos

#define SET_BIT_ATOMIC( var, pos, data ) atomicAnd( ( int* )&var, ~( 1 << pos ) );atomicOr( ( int* )&var, data << pos )

#define Ymask 0x00FF0000
#define Umask 0x0000FF00
#define Vmask 0x000000FF
/* Original trY  */
//#define trY   0x00300000
/* Better value when not using Splines with difusion (Empirical value) */
#define trY 0x00050000
#define trU 0x00000700
#define trV 0x00000006

#define RGBA 0x00000000

#define PIXEL( i, j, widthstep, n_channels ) ( ( j * ( widthstep ) ) + ( i * n_channels ) )

// Retorna posição do bit da aresta da célula vizinha
__device__ int conected_edge( int edge )
{
    return ( 8 - 1 ) - edge;

    /*    switch(edge)
        {
            case 0: return 7;
            case 1: return 6;
            case 2: return 5;
            case 3: return 4;
            case 4: return 3;
            case 5: return 2;
            case 6: return 1;
            case 7: return 0;
        }
    */
}


// Calcula índice da célula vizinha de acordo com a aresta conectada
__device__ int calc_index( int index, int edge, int width )
{
    switch( edge )
    {
        case 0:
            return index + width - 1;

        case 1:
            return index + width;

        case 2:
            return index + width + 1;

        case 3:
            return index - 1;

        case 4:
            return index + 1;

        case 5:
            return index - width - 1;

        case 6:
            return index - width;

        case 7:
            return index - width + 1;
    }

    return 0;
}


/* Initalize RGB to YUV lookup table */
__device__ unsigned int RGBtoYUV( int c )
{
    int r, g, b, y, u, v;

    r = ( c & 0x0000FF );
    g = ( c & 0x00FF00 ) >> 8;
    b = ( c & 0xFF0000 ) >> 16;
    //r = (c & 0xFF0000) >> 16;
    //g = (c & 0x00FF00) >> 8;
    //b = c & 0x0000FF;
    y = int( ( 0.299 * float( r ) + 0.587 * float( g ) + 0.114 * float( b ) ) );
    //u = (unsigned int)(-0.169*r - 0.331*g + 0.5*b) + 128; //Cb
    //v = (unsigned int)(0.5*r - 0.419*g - 0.081*b) + 128;  //Cr
    u = int( ( ( b - y ) * 0.492f ) );
    v = int( ( ( r - y ) * 0.877f ) );


    return ( unsigned int )( ( y << 16 ) + ( u << 8 ) + v );
}


/* Convert value to int type */
__device__ unsigned int DATAtoINT( unsigned int r, unsigned int g, unsigned int b )
{
    unsigned int result;

    /* R_ = xxxx xxxx xxxx xxxx xxxx xxxx RRRR RRRR*/
    unsigned int R_ = ( unsigned int ) r;

    /* G_ = xxxx xxxx xxxx xxxx GGGG GGGG xxxx xxxx*/
    unsigned int G_ = ( unsigned int ) g << 8;

    /* B_ = xxxx xxxx BBBB BBBB xxxx xxxx rrrr rrrr*/
    unsigned int B_ = ( unsigned int ) b << 16;
    //unsigned int A_ = 0x00FFFFFF;


    /* R_ = 0000 0000 0000 0000 0000 0000 RRRR RRRR*/
    R_ &= 0x000000FF;

    /* G_ = 0000 0000 0000 0000 GGGG GGGG 0000 0000*/
    G_ = ( G_ & 0x0000FF00 );

    /* B_ = 0000 0000 BBBB BBBB 0000 0000 0000 0000*/
    B_ = ( B_ & 0x00FF0000 );


    /* result = 0000 0000 BBBB BBBB GGGG GGGG RRRR RRRR*/
    result = ( ( R_ | G_ | B_ ) );

    return result;
}


int __device__ abs_( int a )
{
    if( a < 0 )
    {
        return ( -1 * a );
    }
    else
    {
        return a;
    }
}


__device__ int diff( int w, int h, int pos, int width, int height, int img_widthstep, char* pixel_data )
{
    int result;
    unsigned int YUV1;
    unsigned int YUV2;

    /* Pixels colors in RGB to be converted */
    char* pixel_src, * pixel_dst;

    /* Pixels colors in YUV space to be compared */
    unsigned int pixel_src_YUV, pixel_dst_YUV;

    /* Height and Width position of pixel_dst */
    int n_h, n_w;

    //   +----+----+----+
    //   |    |    |    |
    //   | 0  | 1  | 2  |
    //   +----+----+----+
    //   |    |    |    |
    //   | 3  | x  | 4  |
    //   +----+----+----+
    //   |    |    |    |
    //   | 5  | 6  | 7  |
    //   +----+----+----+

    //    uchar r = img->imageData[h*img->widthStep + w*nChannels];
    //    uchar g = img->imageData[h*img->widthStep + w*nChannels+1];
    //    uchar b = img->imageData[h*img->widthStep + w*nChannels+2];

    pixel_src = &( pixel_data[ PIXEL( w, h, img_widthstep, 3 ) ] );

    // __DEBUG__
//    if ( (w == 2) && (h == 1) )
//    {
//        printf("w: %d, h: %d, pos: %d\n", w, h, pos);
//        printf("width: %d\n", width);
//        printf("n_channels: %d\n", 3);
//        printf("index: %d\n", PIXEL(w, h, img_widthstep, 3));
//        printf("widthstep: %d\n", img_widthstep);
//        printf("pixel_src[0]: %d\n", (unsigned char) pixel_src[0]);
//        printf("pixel_src[1]: %d\n", (unsigned char) pixel_src[1]);
//        printf("pixel_src[2]: %d\n", (unsigned char) pixel_src[2]);
//    }

    pixel_src_YUV = DATAtoINT( pixel_src[ 0 ], pixel_src[ 1 ], pixel_src[ 2 ] );

    switch( pos )
    {
        case 0:
            n_w = ( w - 1 );
            n_h = ( h + 1 );
            break;

        case 1:
            n_w = ( w );
            n_h = ( h + 1 );
            break;

        case 2:
            n_w = ( w + 1 );
            n_h = ( h + 1 );
            break;

        case 3:
            n_w = ( w - 1 );
            n_h = ( h );
            break;

        case 4:
            n_w = ( w + 1 );
            n_h = ( h );
            break;

        case 5:
            n_w = ( w - 1 );
            n_h = ( h - 1 );
            break;

        case 6:
            n_w = ( w );
            n_h = ( h - 1 );
            break;

        case 7:
            n_w = ( w + 1 );
            n_h = ( h - 1 );
            break;
    }


    //printf("pos: %d, n_w: %d, n_h: %d\n", pos, n_w, n_h);


    //index = n_h*img->getWidth() + n_w;

    /* Check if index is inside image boundaries */
    //if ((index > 0) && (index < ((img->getWidth())*(img->getHeight())-1)))
    if( !( ( n_w < 0 ) || ( n_h < 0 ) ||
           ( n_w >= width ) ||
           ( n_h >= height ) ) )
    {
        pixel_dst = &( pixel_data[ PIXEL( n_w, n_h, img_widthstep, 3 ) ] );

        pixel_dst_YUV = DATAtoINT( pixel_dst[ 0 ], pixel_dst[ 1 ], pixel_dst[ 2 ] );

        if( ( w == 1 ) && ( h == 1 ) )
        {
            //printf("pos: %d, pixel_dst[0]: %d, pixel_dst[1]: %d, pixel_dst[2]: %d\n",
            //         pos, (unsigned char) pixel_dst[0], (unsigned char) pixel_dst[1], (unsigned char) pixel_dst[2]);
        }


        /* Mask against RGB_MASK to discard the alpha channel */
        YUV1 = RGBtoYUV( pixel_src_YUV );
        YUV2 = RGBtoYUV( pixel_dst_YUV );

        /* __DEBUG__ */
//        if ((w == 2) && (h==1)){
//            printf("YUV_src: %d, YUV_dest: %d \n",
//                     YUV1,
//                     YUV2);
//            printf("trY: %d, trU: %d, trV: %d \n",
//                     trY,
//                     trU,
//                     trV);
//            printf("maskY_src: %d , maskY_dst: %d, diff: %d, abs_diff: %d\n",
//                     YUV1 & Ymask,
//                     YUV2 & Ymask,
//                     ( ( YUV1 & Ymask ) - ( YUV2 & Ymask ) ),
//                     abs_( ( YUV1 & Ymask ) - ( YUV2 & Ymask ) ) );
//            printf("maskU_src: %d , maskU_dst: %d, diff: %d, abs_diff: %d\n",
//                     YUV1 & Umask,
//                     YUV2 & Umask,
//                     ( ( YUV1 & Umask ) - ( YUV2 & Umask ) ),
//                     abs_((YUV1 & Umask) - (YUV2 & Umask)));
//            printf("maskV_src: %d , maskV_dst: %d, diff: %d, abs_diff: %d\n",
//                     YUV1 & Vmask,
//                     YUV2 & Vmask,
//                     ( ( YUV1 & Vmask ) - ( YUV2 & Vmask ) ),
//                     abs_((YUV1 & Vmask) - (YUV2 & Vmask)));
//        }

        /* cound not use "fabs" function because of weirds results */
        result = ( ( abs_( ( YUV1 & Ymask ) - ( YUV2 & Ymask ) ) > trY ) ||
                   ( abs_( ( YUV1 & Umask ) - ( YUV2 & Umask ) ) > trU ) ||
                   ( abs_( ( YUV1 & Vmask ) - ( YUV2 & Vmask ) ) > trV ) );

        //printf("result: %d \n", result);

        /* Print function data */

        //std::cout << "index src - x: " << w << " y:  " << h << std::endl;
        //std::cout << "index dst - x: " << n_w << " y:  " << n_h << std::endl;
        //std::cout << "pos: " << pos << " - ";
        //std::cout << "results: " << result << std::endl;

        //if (result != 0) {cout << result << endl;}
        return result;
    }
    else
    {
        return 1;
    }
}


__device__ bool crossedEdge( char* graph, int i, int j, int width, int edge )
{
    int index = ( j * width + i );

    int i0 = ( ( j + 1 ) * width + i - 1 );
    int i1 = ( ( j + 1 ) * width + i );
    int i2 = ( ( j + 1 ) * width + i + 1 );
    int i3 = ( ( j ) * width + i - 1 );
    int i4 = ( ( j ) * width + i + 1 );
    int i5 = ( ( j - 1 ) * width + i - 1 );
    int i6 = ( ( j - 1 ) * width + i );
    int i7 = ( ( j - 1 ) * width + i + 1 );

    switch( edge )
    {
        case 0:
            if( ( CHECK_BIT( graph[ i3 ], 2 ) ) &&
                ( CHECK_BIT( graph[ i0 ], 7 ) ) &&
                ( CHECK_BIT( graph[ index ], 0 ) ) &&
                ( CHECK_BIT( graph[ i1 ], 5 ) ) )
            {
                return true;
            }

        case 2:
            if( ( CHECK_BIT( graph[ index ], 2 ) ) &&
                ( CHECK_BIT( graph[ i1 ], 7 ) ) &&
                ( CHECK_BIT( graph[ i4 ], 0 ) ) &&
                ( CHECK_BIT( graph[ i2 ], 5 ) ) )
            {
                return true;
            }

        case 7:
            if( ( CHECK_BIT( graph[ i5 ], 2 ) ) &&
                ( CHECK_BIT( graph[ i3 ], 7 ) ) &&
                ( CHECK_BIT( graph[ i6 ], 0 ) ) &&
                ( CHECK_BIT( graph[ index ], 5 ) ) )
            {
                return true;
            }

        case 5:
            if( ( CHECK_BIT( graph[ i6 ], 2 ) ) &&
                ( CHECK_BIT( graph[ index ], 7 ) ) &&
                ( CHECK_BIT( graph[ i7 ], 0 ) ) &&
                ( CHECK_BIT( graph[ i4 ], 5 ) ) )
            {
                return true;
            }
    }

    return false;
}


__device__ bool checkValence2EdgeGPU( char* graph, int i_1, int j_1, int i_2, int j_2, int width, int edge_1,
                                      int edge_2, char nodeResult )
{
    //int k_n;     // last conected edge
    int sum_1 = 0;
    int sum_2 = 0; // soma de arestas conectadas além da 2
    int index_1 = ( j_1 * width + i_1 );
    int index_2 = ( j_2 * width + i_2 );

    char node_1 = graph[ index_1 ];

    if( nodeResult != 255 )
    {
        node_1 = nodeResult;
    }

    for( int k = ( edge_1 + 1 ); k < ( edge_1 + 8 ); k++ )
    {
        if( CHECK_BIT( node_1, k % 8 ) )
        {
            //if (!crossedEdge(graph, i_1, j_1, width, k%8))
            //{
            sum_1++;
            //k_n = k;
            //}
        }
    }
    for( int k = ( edge_2 + 1 ); k < ( edge_2 + 8 ); k++ )
    {
        if( CHECK_BIT( graph[ index_2 ], k % 8 ) )
        {
            //if (!crossedEdge(graph, i_2, j_2, width, k%8))
            //{
            sum_2++;
            //k_n = k;
            //}
        }
    }
    //cout << "sum_1: " << sum_1 << endl;
    //cout << "sum_2: " << sum_2 << endl;
    if( ( sum_1 == 1 ) && ( sum_2 == 1 ) )
    {
        return true;
    }
    else
    {
        return false;
    }
}


__device__ bool checkValence2Edge( char* graph, int index_1, int index_2, int edge_1, int edge_2 )
{
    //int k_n;     // last conected edge
    int sum_1 = 0;
    int sum_2 = 0; // soma de arestas conectadas além da 2

    for( int k = ( edge_1 + 1 ); k < ( edge_1 + 8 ); k++ )
    {
        if( CHECK_BIT( graph[ index_1 ], k % 8 ) )
        {
            sum_1++;
            //k_n = k;
        }
    }
    for( int k = ( edge_2 + 1 ); k < ( edge_2 + 8 ); k++ )
    {
        if( CHECK_BIT( graph[ index_2 ], k % 8 ) )
        {
            sum_2++;
            //k_n = k;
        }
    }
    //cout << "sum_1: " << sum_1 << endl;
    //cout << "sum_2: " << sum_2 << endl;
    if( ( sum_1 == 1 ) && ( sum_2 == 1 ) )
    {
        return true;
    }
    else
    {
        return false;
    }
}


/* Varre o nó para saber se existe algum outro bit ativo
   Retorna true se apenas edge_1 está ativo             */
__device__ bool checkValence2Vertex( char* graph, int index_1, int edge_1 )
{
    //int k_n;     // last conected edge
    int sum = 0; // soma de arestas conectadas além da 2
    for( int k = ( edge_1 + 1 ); k < ( edge_1 + 8 ); k++ )
    {
        if( CHECK_BIT( graph[ index_1 ], k % 8 ) )
        {
            sum++;
            //k_n = k;
        }
    }
    //cout << sum << endl;
    if( sum == 0 )
    {
        return true;
    }
    else
    {
        return false;
    }
}


/* Varre o nó para saber se existe algum outro bit ativo
   Retorna true quando mais um (e apenas um) link estiver atvo            */
__device__ bool checkVertexV2( char* graph, int index_1, int edge_1 )
{
    //int k_n;     // last conected edge
    int sum = 0; // soma de arestas conectadas além da 2
    for( int k = ( edge_1 + 1 ); k < ( edge_1 + 8 ); k++ )
    {
        if( CHECK_BIT( graph[ index_1 ], k % 8 ) )
        {
            sum++;
            //k_n = k;
        }
    }
    //cout << sum << endl;
    if( sum == 1 )
    {
        return true;
    }
    else
    {
        return false;
    }
}


__device__ bool checkValence2VertexGPU( char* graph, int index_1, int edge_1, char nodeResult )
{
    //int k_n;     // last conected edge
    int sum = 0; // soma de arestas conectadas além da 2

    char node_1 = graph[ index_1 ];

    if( nodeResult != 255 )
    {
        node_1 = nodeResult;
    }

    for( int k = ( edge_1 + 1 ); k < ( edge_1 + 8 ); k++ )
    {
        if( CHECK_BIT( node_1, k % 8 ) )
        {
            sum++;
            //k_n = k;
        }
    }
    //cout << sum << endl;
    if( sum == 0 )
    {
        return true;
    }
    else
    {
        return false;
    }
}


/* Calculate the size of a valence-2 path starting prom index */
__device__ int calcVal2PathSize( char* graph, int index, int edge, int& result, int width )
{
    int k_n; // Outra aresta quando o segmento é de valência 2
    int sum = 0;
    //switch (side){
    //    case 0: {
    // a partir de edge + 1 (edge já é conexo)
    for( int k = ( edge + 1 ); k < ( edge + 8 ); k++ )
    {
        if( CHECK_BIT( graph[ index ], k % 8 ) )
        {
            sum++;
            k_n = k;
        }
    }
    // Compare to a big number to not enter in loop
    if( result > 30 )
    {
        return result;
    }
    if( sum == 1 )
    {
        result++;
        calcVal2PathSize( graph, calc_index( index, k_n % 8, width ), conected_edge( k_n % 8 ), result, width );
    }
    else if( sum == 0 )
    {
        return result;
    }
    return result;
    //}
    //}
}


__device__ void processHeuristics( char* graph, int i, int j, int width, int height )
{
    int seg_size = 0;
    int seg_size2 = 0;

    int i1 = ( j * width + i );
    int i2 = ( ( j + 1 ) * width + i );
    int i3 = ( j * width + i + 1 );
    int i4 = ( ( j + 1 ) * width + i + 1 );



    // Both edges are
    if( ( checkValence2Edge( graph, i1, i4, 2, 5 ) ) && ( checkValence2Edge( graph, i3, i2, 0, 7 ) ) )
    {
    }
    else
    {
        /* Just i1-i4 edge is valence 2 */
        /*  i2  /i4
               /
            i1/  i3  */
        if( checkValence2Edge( graph, i1, i4, 2, 5 ) )
        {
            /* Remove i2-i3 edge */
            SET_BIT( graph[ i2 ], 7, 0 );
            SET_BIT( graph[ i3 ], 0, 0 );
        }
        else
        /* Just i2-i3 edge is valence 2 */
        /*  i2\  i4
               \
            i1  \i3  */
        if( checkValence2Edge( graph, i3, i2, 0, 7 ) )
        {
            /* Remove i1-i4 edge */
            SET_BIT( graph[ i1 ], 2, 0 );
            SET_BIT( graph[ i4 ], 5, 0 );
        }
        else
        /* Island Heuristic */
        if( ( ( checkValence2Vertex( graph, i1, 2 ) ) || ( checkValence2Vertex( graph, i4, 5 ) ) ) &&
            ( !( checkValence2Vertex( graph, i3, 0 ) ) ) && ( !( checkValence2Vertex( graph, i2, 7 ) ) ) )
        {
            SET_BIT( graph[ i2 ], 7, 0 );
            SET_BIT( graph[ i3 ], 0, 0 );
        }
        else
        /* Island Heuristic */
        if( ( checkValence2Vertex( graph, i3, 0 ) ) || ( checkValence2Vertex( graph, i2, 7 ) ) &&
            ( !( checkValence2Vertex( graph, i1, 2 ) ) ) && ( !( checkValence2Vertex( graph, i4, 5 ) ) ) )
        {
            SET_BIT( graph[ i1 ], 2, 0 );
            SET_BIT( graph[ i4 ], 5, 0 );
        }
        else
        {
            // Calcula tamanho partindo da aresta 2 de i1 e partindo da aresta 5 de i4. Depois soma

            //usleep(5000 * 1000);
            calcVal2PathSize( graph, i1, 2, seg_size, width );

            calcVal2PathSize( graph, i4, 5, seg_size, width );
            //cout << "i1 seg_size total: " << seg_size << endl;


            // Calcula tamanho partindo da aresta 0 de i3 e partindo de 7 da aresta i2. Depois soma

            //usleep(5000 * 1000);
            calcVal2PathSize( graph, i3, 0, seg_size2, width );

            calcVal2PathSize( graph, i2, 7, seg_size2, width );
            //cout << "  i3 seg_size total: " << seg_size2 << endl;

            // compara tamanho de cada curva e retira aresta da menor
            if( seg_size2 < seg_size )
            {
                //cout << "seg_size maior retira aresta 7 e 0" << endl;
                SET_BIT( graph[ i2 ], 7, 0 );
                SET_BIT( graph[ i3 ], 0, 0 );
            }
            else
            {
                //cout << "seg_size2 maior retira aresta 2 e 5" << endl;
                SET_BIT( graph[ i1 ], 2, 0 );
                SET_BIT( graph[ i4 ], 5, 0 );
            }
        }
    }
}


__device__ int processHeuristics22( char* graph, int i, int j, int width, int height, char nodeResult )
{
    int seg_size = 0;
    int seg_size2 = 0;

    int i1 = ( j * width + i );
    int i2 = ( ( j + 1 ) * width + i );
    int i3 = ( j * width + i + 1 );
    int i4 = ( ( j + 1 ) * width + i + 1 );


    // Both edges are
//    if ((checkValence2Edge(graph, i1, i4, 2, 5)) && (checkValence2Edge(graph, i3, i2, 0, 7)))
//    {

//    } else {
    /* Just i1-i4 edge is valence 2 */
    /*  i2  /i4
           /
        i1/  i3  */
    //if (checkValence2Edge(graph, i1, i4, 2, 5)){
    if( checkValence2EdgeGPU( graph, i, j, i + 1, j + 1, width, 2, 5, nodeResult ) )
    {
        /* Remove i2-i3 edge */
        return false;
    }
    else
    /* Just i2-i3 edge is valence 2 */
    /*  i2\  i4
           \
        i1  \i3  */
    //if(checkValence2Edge(graph, i3, i2, 0, 7)){
    if( checkValence2EdgeGPU( graph, i + 1, j, i, j + 1, width, 0, 7, 255 ) )
    {
        /* Remove i1-i4 edge */
        return true;
    }
    else
    /* Island Heuristic */
    if( ( ( checkValence2VertexGPU( graph, i1, 2, nodeResult ) ) || ( checkValence2Vertex( graph, i4, 5 ) ) ) &&
        ( !( checkValence2Vertex( graph, i3, 0 ) ) ) && ( !( checkValence2Vertex( graph, i2, 7 ) ) ) )
    {
        return false;
    }
    else
    /* Island Heuristic */
    if( ( checkValence2Vertex( graph, i3, 0 ) ) || ( checkValence2Vertex( graph, i2, 7 ) ) &&
        ( !( checkValence2VertexGPU( graph, i1, 2, nodeResult ) ) ) && ( !( checkValence2Vertex( graph, i4, 5 ) ) ) )
    {
        return true;
    }
    else
    {
        // Calcula tamanho partindo da aresta 2 de i1 e partindo da aresta 5 de i4. Depois soma

        //usleep(5000 * 1000);
        calcVal2PathSize( graph, i1, 2, seg_size, width );

        calcVal2PathSize( graph, i4, 5, seg_size, width );
        //cout << "i1 seg_size total: " << seg_size << endl;


        // Calcula tamanho partindo da aresta 0 de i3 e partindo de 7 da aresta i2. Depois soma

        //usleep(5000 * 1000);
        calcVal2PathSize( graph, i3, 0, seg_size2, width );

        calcVal2PathSize( graph, i2, 7, seg_size2, width );
        //cout << "  i3 seg_size total: " << seg_size2 << endl;

        // compara tamanho de cada curva e retira aresta da menor
        if( seg_size2 < seg_size )
        {
            //cout << "seg_size maior retira aresta 7 e 0" << endl;
            return false;
        }
        else
        {
            //cout << "seg_size2 maior retira aresta 2 e 5" << endl;
            return true;
        }


        // Link can't be solved with this four nodes
        //if ( (seg_size == 0) && (seg_size2 == 0) )
        //{
        //crossCheck_Neighbors(graph, i, j, width, height);
        //}
    }
    //}
}


__device__ int processHeuristics2( char* graph, int i, int j, int width, int height )
{
    int seg_size = 0;
    int seg_size2 = 0;

    int i1 = ( j * width + i );
    int i2 = ( ( j + 1 ) * width + i );
    int i3 = ( j * width + i + 1 );
    int i4 = ( ( j + 1 ) * width + i + 1 );


    // Both edges are
//    if ((checkValence2Edge(graph, i1, i4, 2, 5)) && (checkValence2Edge(graph, i3, i2, 0, 7)))
//    {

//    } else {
    /* Just i1-i4 edge is valence 2 */
    /*  i2  /i4
           /
        i1/  i3  */
    if( checkValence2Edge( graph, i1, i4, 2, 5 ) )
    {
        /* Remove i2-i3 edge */
        return false;
    }
    else
    /* Just i2-i3 edge is valence 2 */
    /*  i2\  i4
           \
        i1  \i3  */
    if( checkValence2Edge( graph, i3, i2, 0, 7 ) )
    {
        /* Remove i1-i4 edge */
        return true;
    }
    else
    /* Island Heuristic */
    if( ( ( checkValence2Vertex( graph, i1, 2 ) ) || ( checkValence2Vertex( graph, i4, 5 ) ) ) &&
        ( !( checkValence2Vertex( graph, i3, 0 ) ) ) && ( !( checkValence2Vertex( graph, i2, 7 ) ) ) )
    {
        return false;
    }
    else
    /* Island Heuristic */
    if( ( checkValence2Vertex( graph, i3, 0 ) ) || ( checkValence2Vertex( graph, i2, 7 ) ) &&
        ( !( checkValence2Vertex( graph, i1, 2 ) ) ) && ( !( checkValence2Vertex( graph, i4, 5 ) ) ) )
    {
        return true;
    }
    else
    {
        // Calcula tamanho partindo da aresta 2 de i1 e partindo da aresta 5 de i4. Depois soma

        //usleep(5000 * 1000);
        calcVal2PathSize( graph, i1, 2, seg_size, width );

        calcVal2PathSize( graph, i4, 5, seg_size, width );
        //cout << "i1 seg_size total: " << seg_size << endl;


        // Calcula tamanho partindo da aresta 0 de i3 e partindo de 7 da aresta i2. Depois soma

        //usleep(5000 * 1000);
        calcVal2PathSize( graph, i3, 0, seg_size2, width );

        calcVal2PathSize( graph, i2, 7, seg_size2, width );
        //cout << "  i3 seg_size total: " << seg_size2 << endl;

        // compara tamanho de cada curva e retira aresta da menor
        if( seg_size2 < seg_size )
        {
            //cout << "seg_size maior retira aresta 7 e 0" << endl;
            return false;
        }
        else
        {
            //cout << "seg_size2 maior retira aresta 2 e 5" << endl;
            return true;
        }


        // Link can't be solved with this four nodes
        //if( ( seg_size == 0 ) && ( seg_size2 == 0 ) )
        //{
            //crossCheck_Neighbors(graph, i, j, width, height);
        //}
    }
    //}
}


//__device__ int processHeuristicsNeighbors(char* graph, int i, int j, int width, int height){
//    int seg_size = 0;
//    int seg_size2 = 0;

//    int i1 = (j*width + i);
//    int i2 = ((j+1)*width + i);
//    int i3 = (j*width + i+1);
//    int i4 = ((j+1)*width + i+1);

//    int i0 = ((j+1)*width + i-1);
//    int i1 = ((j+1)*width + i);
//    int i2 = ((j+1)*width + i+1);
//    int i3 = ((j)*width + i-1);
//    int i4 = ((j)*width + i+1);
//    int i5 = ((j-1)*width + i-1);
//    int i6 = ((j-1)*width + i);
//    int i7 = ((j-1)*width + i+1);


//    // Both edges are
////    if ((checkValence2Edge(graph, i1, i4, 2, 5)) && (checkValence2Edge(graph, i3, i2, 0, 7)))
////    {

////    } else {
//        /* Just i1-i4 edge is valence 2 */
//        /*  i2  /i4
//               /
//            i1/  i3  */
//        if (checkValence2Edge(graph, i1, i4, 2, 5)){
//            /* Remove i2-i3 edge */
//            return false;
//        } else
//            /* Just i2-i3 edge is valence 2 */
//            /*  i2\  i4
//                   \
//                i1  \i3  */
//            if(checkValence2Edge(graph, i3, i2, 0, 7)){
//                /* Remove i1-i4 edge */
//                return true;
//            } else
//                /* Island Heuristic */
//                if( ( (checkValence2Vertex(graph, i1, 2)) || (checkValence2Vertex(graph, i4, 5)) ) &&
//                        (!(checkValence2Vertex(graph, i3, 0))) && (!(checkValence2Vertex(graph, i2, 7))) ){
//                    return false;
//                } else
//                    /* Island Heuristic */
//                    if( (checkValence2Vertex(graph, i3, 0)) || (checkValence2Vertex(graph, i2, 7)) &&
//                            (!(checkValence2Vertex(graph, i1, 2))) && (!(checkValence2Vertex(graph, i4, 5))) ){
//                        return true;
//                    } else {

//            // Calcula tamanho partindo da aresta 2 de i1 e partindo da aresta 5 de i4. Depois soma

//            //usleep(5000 * 1000);
//            calcVal2PathSize(graph, i1, 2, seg_size, width);

//            calcVal2PathSize(graph, i4, 5, seg_size, width);
//            //cout << "i1 seg_size total: " << seg_size << endl;


//            // Calcula tamanho partindo da aresta 0 de i3 e partindo de 7 da aresta i2. Depois soma

//            //usleep(5000 * 1000);
//            calcVal2PathSize(graph, i3, 0, seg_size2, width);

//            calcVal2PathSize(graph, i2, 7, seg_size2, width);
//            //cout << "  i3 seg_size total: " << seg_size2 << endl;

//            // compara tamanho de cada curva e retira aresta da menor
//            if (seg_size2 < seg_size)
//            {
//                //cout << "seg_size maior retira aresta 7 e 0" << endl;
//                return false;
//            } else
//            {
//                //cout << "seg_size2 maior retira aresta 2 e 5" << endl;
//                return true;
//            }
//        }
//    //}

//}

//__device__ char crossCheck_Neighbors(char* graph, int width, int height, int i, int j)
//{
//    char node = graph[index];
//    int x = i; int y = j;

//    int index = (y*width + x);

//    int i0 = ((j+1)*width + i-1);
//    int i1 = ((j+1)*width + i);
//    int i2 = ((j+1)*width + i+1);
//    int i3 = ((j)*width + i-1);
//    int i4 = ((j)*width + i+1);
//    int i5 = ((j-1)*width + i-1);
//    int i6 = ((j-1)*width + i);
//    int i7 = ((j-1)*width + i+1);



//    if ( (CHECK_BIT(graph[i3],2)) &&
//         (CHECK_BIT(graph[i0],7)) &&
//         (CHECK_BIT(graph[index],0)) &&
//         (CHECK_BIT(graph[i1],5)) )
//    {
//        if (!(processHeuristics2(graph, i-1, j, width, height)) )
//            SET_BIT(node, 0, 0);
//    }

//    if ( (CHECK_BIT(graph[i5],2)) &&
//         (CHECK_BIT(graph[i3],7)) &&
//         (CHECK_BIT(graph[i6],0)) &&
//         (CHECK_BIT(graph[index],5)) )
//    {
//        if (processHeuristics2(graph, i-1, j-1, width, height))
//            SET_BIT(node, 5, 0);
//    }

//    if ( (CHECK_BIT(graph[i6],2)) &&
//         (CHECK_BIT(graph[index],7)) &&
//         (CHECK_BIT(graph[i7],0)) &&
//         (CHECK_BIT(graph[i4],5)) )
//    {
//        if (!(processHeuristics2(graph, i, j-1, width, height)) )
//            SET_BIT(node, 7, 0);
//    }

//    if ( (CHECK_BIT(graph[index],2)) &&
//         (CHECK_BIT(graph[i1],7)) &&
//         (CHECK_BIT(graph[i4],0)) &&
//         (CHECK_BIT(graph[i2],5)) )
//    {
//        if (processHeuristics2(graph, i, j, width, height))
//            SET_BIT(node, 2, 0);
//    }

//    return node;
//}

/* Verifica ambiguidade de aresta cruzada nos quatro vizinhos da diagonal e tira só as arestas do nó interior "index" */
/* Maneira de resolver problema de concorrência */
__device__ void processHeuristicsCases( char* graph, int j, int i, int width, int height )
{
    //   +----+----+----+
    //   |    |    |    |
    //   | i2 |    | i3 |
    //   +----+----+----+
    //   |    |    |    |
    //   |    | i  |    |
    //   +----+----+----+
    //   |    |    |    |
    //   | i1 |    | i4 |
    //   +----+----+----+

    //int index = ( ( i ) * width - j );
    //int i1 = ((i - 1) * width + (j - 1));
    //int i2 = ((i + 1) * width + (j - 1));
    //int i3 = ((i - 1) * width + (j + 1));
    //int i4 = ((i + 1) * width + (j + 1));

    //if (processHeuristics2(graph, j-1, i-1, width, height))
    //    SET_BIT(index, 5, 0);
    //if (processHeuristics2(graph, j-1, i+1, width, height))
    //    SET_BIT(index, 0, 0);
    //if (processHeuristics2(graph, j+1, i-1, width, height))
    //    SET_BIT(index, 2, 0);
    //if (processHeuristics2(graph, j+1, i+1, width, height))
    //    SET_BIT(index, 7, 0);
}


/* Change edges according to heuristics */
__device__ void crossCheck( char* graph, int width, int height, int i, int j )
{
    for( int i = 0; i < height; i++ )
    {
        for( int j = 0; j < width; j++ )
        {
            /*    i2 --- i4
                  |      |
                  i1 --- i3    */

            int i1 = ( i * width + j );
            int i2 = ( ( i + 1 ) * width + j );
            int i3 = ( i * width + j + 1 );
            int i4 = ( ( i + 1 ) * width + j + 1 );

            // Checa se blocos 2x2 estão totalmente conectados

            /* Checa se blocos 2x2 estão totalmente conectados */

            /*  3--- i2 ---(4)   (3)--- i4 ---4
                    / | \              / | \
                   /  |  \            /  |  \
                  5  (6)  7          5  (6)  7


                  0  (1)   2         0  (1)   2
                   \  |   /           \  |   /
                    \ |  /             \ |  /
                 3--- i1 ---(4)   (3)--- i3 ---4         */

            if( ( CHECK_BIT( graph[ i1 ], 1 ) ) && ( CHECK_BIT( graph[ i1 ], 4 ) ) &&
                ( CHECK_BIT( graph[ i2 ], 6 ) ) && ( CHECK_BIT( graph[ i2 ], 4 ) ) &&
                ( CHECK_BIT( graph[ i3 ], 3 ) ) && ( CHECK_BIT( graph[ i3 ], 1 ) ) &&
                ( CHECK_BIT( graph[ i4 ], 6 ) ) && ( CHECK_BIT( graph[ i4 ], 3 ) ) )
            {
                //cvCircle( graph_img, cvPoint(n_j+half_sg,n_i+half_sg), 3, CV_RGB( 0,  0, 0 ), CV_FILLED, 8, 0 );
                //remove arestas cruzadas
                SET_BIT( graph[ i1 ], 2, 0 );
                SET_BIT( graph[ i2 ], 7, 0 );
                SET_BIT( graph[ i3 ], 0, 0 );
                SET_BIT( graph[ i4 ], 5, 0 );
            }
            // Checa pelas arestas cruzadas

            /*  3--- i2 ---4        3--- i4 ---4
                    / | \              / | \
                   /  |  \            /  |  \
                  5   6   (7)       (5)  6   7


                  0   1   (2)       (0)  1    2
                   \  |   /           \  |   /
                    \ |  /             \ |  /
                 3--- i1 ---4       3--- i3 ---4         */

            if( ( CHECK_BIT( graph[ i1 ], 2 ) ) &&
                ( CHECK_BIT( graph[ i2 ], 7 ) ) &&
                ( CHECK_BIT( graph[ i3 ], 0 ) ) &&
                ( CHECK_BIT( graph[ i4 ], 5 ) ) )
            {
                processHeuristics( graph, j, i, width, height );
            }
        }
    }
}


/* Change edges according to heuristics */
__device__ void crossCheck_2( char* graph, int width, int height, int i, int j )
{
    /*    i2 --- i4
          |      |
          i1 --- i3    */

    int i1 = ( j * width + i );
    int i2 = ( ( j + 1 ) * width + i );
    int i3 = ( j * width + i + 1 );
    int i4 = ( ( j + 1 ) * width + i + 1 );

    // Checa se blocos 2x2 estão totalmente conectados

    /* Checa se blocos 2x2 estão totalmente conectados */

    /*  3--- i2 ---(4)   (3)--- i4 ---4
            / | \              / | \
           /  |  \            /  |  \
          5  (6)  7          5  (6)  7


          0  (1)   2         0  (1)   2
           \  |   /           \  |   /
            \ |  /             \ |  /
         3--- i1 ---(4)   (3)--- i3 ---4         */

    if( ( CHECK_BIT( graph[ i1 ], 1 ) ) && ( CHECK_BIT( graph[ i1 ], 4 ) ) &&
        ( CHECK_BIT( graph[ i2 ], 6 ) ) && ( CHECK_BIT( graph[ i2 ], 4 ) ) &&
        ( CHECK_BIT( graph[ i3 ], 3 ) ) && ( CHECK_BIT( graph[ i3 ], 1 ) ) &&
        ( CHECK_BIT( graph[ i4 ], 6 ) ) && ( CHECK_BIT( graph[ i4 ], 3 ) ) )
    {
        //remove arestas cruzadas
        SET_BIT( graph[ i1 ], 2, 0 );
        SET_BIT( graph[ i2 ], 7, 0 );
        SET_BIT( graph[ i3 ], 0, 0 );
        SET_BIT( graph[ i4 ], 5, 0 );
    }
}


/* Change edges according to heuristics */
__device__ void crossCheck_3( char* graph, int width, int height, int i, int j )
{
    /*    i2 --- i4
          |      |
          i1 --- i3    */

    int i1 = ( j * width + i );
    int i2 = ( ( j + 1 ) * width + i );
    int i3 = ( j * width + i + 1 );
    int i4 = ( ( j + 1 ) * width + i + 1 );

    int i5 = ( ( j + 1 ) * width + i );
    int i6 = ( ( j + 2 ) * width + i );
    int i7 = ( ( j + 1 ) * width + i + 1 );
    int i8 = ( ( j + 2 ) * width + i + 1 );

    int i9 = ( j * width + i + 1 );
    int i10 = ( ( j + 1 ) * width + i + 1 );
    int i11 = ( j * width + i + 2 );
    int i12 = ( ( j + 1 ) * width + i + 2 );

    int i13 = ( ( j + 1 ) * width + i + 1 );
    int i14 = ( ( j + 2 ) * width + i + 1 );
    int i15 = ( ( j + 1 ) * width + i + 2 );
    int i16 = ( ( j + 2 ) * width + i + 2 );

    // Checa se blocos 2x2 estão totalmente conectados

    /* Checa se blocos 2x2 estão totalmente conectados */

    /*  3--- i2 ---(4)   (3)--- i4 ---4
            / | \              / | \
           /  |  \            /  |  \
          5  (6)  7          5  (6)  7


          0  (1)   2         0  (1)   2
           \  |   /           \  |   /
            \ |  /             \ |  /
         3--- i1 ---(4)   (3)--- i3 ---4         */

    if( ( CHECK_BIT( graph[ i1 ], 1 ) ) && ( CHECK_BIT( graph[ i1 ], 4 ) ) &&
        ( CHECK_BIT( graph[ i2 ], 6 ) ) && ( CHECK_BIT( graph[ i2 ], 4 ) ) &&
        ( CHECK_BIT( graph[ i3 ], 3 ) ) && ( CHECK_BIT( graph[ i3 ], 1 ) ) &&
        ( CHECK_BIT( graph[ i4 ], 6 ) ) && ( CHECK_BIT( graph[ i4 ], 3 ) ) )
    {
        //remove arestas cruzadas
        SET_BIT( graph[ i4 ], 5, 0 );
    }
    if( ( CHECK_BIT( graph[ i5 ], 1 ) ) && ( CHECK_BIT( graph[ i5 ], 4 ) ) &&
        ( CHECK_BIT( graph[ i6 ], 6 ) ) && ( CHECK_BIT( graph[ i6 ], 4 ) ) &&
        ( CHECK_BIT( graph[ i7 ], 3 ) ) && ( CHECK_BIT( graph[ i7 ], 1 ) ) &&
        ( CHECK_BIT( graph[ i8 ], 6 ) ) && ( CHECK_BIT( graph[ i8 ], 3 ) ) )
    {
        //remove arestas cruzadas
        SET_BIT( graph[ i6 ], 7, 0 );
    }
    if( ( CHECK_BIT( graph[ i9 ], 1 ) ) && ( CHECK_BIT( graph[ i9 ], 4 ) ) &&
        ( CHECK_BIT( graph[ i10 ], 6 ) ) && ( CHECK_BIT( graph[ i10 ], 4 ) ) &&
        ( CHECK_BIT( graph[ i11 ], 3 ) ) && ( CHECK_BIT( graph[ i11 ], 1 ) ) &&
        ( CHECK_BIT( graph[ i12 ], 6 ) ) && ( CHECK_BIT( graph[ i12 ], 3 ) ) )
    {
        //remove arestas cruzadas
        SET_BIT( graph[ i11 ], 0, 0 );
    }
    if( ( CHECK_BIT( graph[ i13 ], 1 ) ) && ( CHECK_BIT( graph[ i13 ], 4 ) ) &&
        ( CHECK_BIT( graph[ i14 ], 6 ) ) && ( CHECK_BIT( graph[ i14 ], 4 ) ) &&
        ( CHECK_BIT( graph[ i15 ], 3 ) ) && ( CHECK_BIT( graph[ i15 ], 1 ) ) &&
        ( CHECK_BIT( graph[ i16 ], 6 ) ) && ( CHECK_BIT( graph[ i16 ], 3 ) ) )
    {
        //remove arestas cruzadas
        SET_BIT( graph[ i6 ], 5, 0 );
    }
}


/* Change edges according to heuristics */
__device__ void crossCheck_4( char* graph, int width, int height, int i, int j )
{
    int index = ( j * width + i );

    int i0 = ( ( j - 1 ) * width + i - 1 );
    int i1 = ( ( j ) * width + i - 1 );
    int i2 = ( ( j + 1 ) * width + i - 1 );
    int i3 = ( ( j - 1 ) * width + i );
    int i4 = ( ( j + 1 ) * width + i );
    int i5 = ( ( j - 1 ) * width + i + 1 );
    int i6 = ( ( j ) * width + i + 1 );
    int i7 = ( ( j + 1 ) * width + i + 1 );

    // Checa se blocos 2x2 estão totalmente conectados

    /* Checa se blocos 2x2 estão totalmente conectados */

    /*  3--- i2 ---(4)   (3)--- i4 ---4
            / | \              / | \
           /  |  \            /  |  \
          5  (6)  7          5  (6)  7


          0  (1)   2         0  (1)   2
           \  |   /           \  |   /
            \ |  /             \ |  /
         3--- i1 ---(4)   (3)--- i3 ---4         */

    if( ( CHECK_BIT( graph[ i1 ], 1 ) ) && ( CHECK_BIT( graph[ i1 ], 4 ) ) &&
        ( CHECK_BIT( graph[ i2 ], 6 ) ) && ( CHECK_BIT( graph[ i2 ], 4 ) ) &&
        ( CHECK_BIT( graph[ index ], 3 ) ) && ( CHECK_BIT( graph[ index ], 1 ) ) &&
        ( CHECK_BIT( graph[ i4 ], 6 ) ) && ( CHECK_BIT( graph[ i4 ], 3 ) ) )
    {
        //remove arestas cruzadas
        SET_BIT( graph[ index ], 0, 0 );
    }

    if( ( CHECK_BIT( graph[ index ], 1 ) ) && ( CHECK_BIT( graph[ index ], 4 ) ) &&
        ( CHECK_BIT( graph[ i4 ], 6 ) ) && ( CHECK_BIT( graph[ i4 ], 4 ) ) &&
        ( CHECK_BIT( graph[ i6 ], 3 ) ) && ( CHECK_BIT( graph[ i6 ], 1 ) ) &&
        ( CHECK_BIT( graph[ i7 ], 6 ) ) && ( CHECK_BIT( graph[ i7 ], 3 ) ) )
    {
        //remove arestas cruzadas
        SET_BIT( graph[ index ], 2, 0 );
    }

    if( ( CHECK_BIT( graph[ i0 ], 1 ) ) && ( CHECK_BIT( graph[ i0 ], 4 ) ) &&
        ( CHECK_BIT( graph[ i1 ], 6 ) ) && ( CHECK_BIT( graph[ i1 ], 4 ) ) &&
        ( CHECK_BIT( graph[ i3 ], 3 ) ) && ( CHECK_BIT( graph[ i3 ], 1 ) ) &&
        ( CHECK_BIT( graph[ index ], 6 ) ) && ( CHECK_BIT( graph[ index ], 3 ) ) )
    {
        //remove arestas cruzadas
        SET_BIT( graph[ index ], 5, 0 );
    }

    if( ( CHECK_BIT( graph[ i3 ], 1 ) ) && ( CHECK_BIT( graph[ i3 ], 4 ) ) &&
        ( CHECK_BIT( graph[ index ], 6 ) ) && ( CHECK_BIT( graph[ index ], 4 ) ) &&
        ( CHECK_BIT( graph[ i5 ], 3 ) ) && ( CHECK_BIT( graph[ i5 ], 1 ) ) &&
        ( CHECK_BIT( graph[ i6 ], 6 ) ) && ( CHECK_BIT( graph[ i6 ], 3 ) ) )
    {
        //remove arestas cruzadas
        SET_BIT( graph[ index ], 7, 0 );
    }
}


/* Return true if all nodes of the crossing edges are not valence 2 */
__device__ bool check_neighboord( char* graph, int i, int j, int width )
{
    /*  i2   i4

        i1   i3  */

    int i1 = ( j * width + i );
    int i2 = ( ( j + 1 ) * width + i );
    int i3 = ( j * width + i + 1 );
    int i4 = ( ( j + 1 ) * width + i + 1 );

    if( !checkVertexV2( graph, i1, 2 )
        && !checkVertexV2( graph, i2, 7 )
        && !checkVertexV2( graph, i3, 0 )
        && !checkVertexV2( graph, i4, 5 ) )
    {
        return true;
    }
    else
    {
        return false;
    }
}


/**
 * @brief processHeuristics             Process heuristics to improve graph result
 * @param j                             Coordinate j - x axis
 * @param i                             Coordinate i - y axis
 */
__device__ void processHeuristicsWindow( char* matrix, int i, int j, int width )
{
    int seg_size = 0;
    int seg_size2 = 0;

    int i1 = ( j * width + i );
    int i2 = ( ( j + 1 ) * width + i );
    int i3 = ( j * width + i + 1 );
    int i4 = ( ( j + 1 ) * width + i + 1 );

    // Both edges are
//    if ((checkValence2Edge(i1, i4, 2, 5)) && (checkValence2Edge(i3, i2, 0, 7)))
//    {

//    } else {
    /* Just i1-i4 edge is valence 2 */
    /*  i2  /i4
           /
        i1/  i3  */
    if( ( checkValence2Edge( matrix, i1, i4, 2, 5 ) ) && !( checkValence2Edge( matrix, i3, i2, 0, 7 ) ) )
    {
        /* Remove i2-i3 edge */
        SET_BIT( matrix[ i2 ], 7, 0 );
        SET_BIT( matrix[ i3 ], 0, 0 );
    }
    else
    /* Just i2-i3 edge is valence 2 */
    /*  i2\  i4
           \
        i1  \i3  */
    if( ( checkValence2Edge( matrix, i3, i2, 0, 7 ) ) && !( checkValence2Edge( matrix, i1, i4, 2, 5 ) ) )
    {
        /* Remove i1-i4 edge */
        SET_BIT( matrix[ i1 ], 2, 0 );
        SET_BIT( matrix[ i4 ], 5, 0 );
    }
    else
    /* Island Heuristic */
    if( ( ( checkValence2Vertex( matrix, i1, 2 ) ) || ( checkValence2Vertex( matrix, i4, 5 ) ) ) &&
        ( !( checkValence2Vertex( matrix, i3, 0 ) ) ) && ( !( checkValence2Vertex( matrix, i2, 7 ) ) ) )
    {
        SET_BIT( matrix[ i2 ], 7, 0 );
        SET_BIT( matrix[ i3 ], 0, 0 );
    }
    else
    /* Island Heuristic */
    if( ( checkValence2Vertex( matrix, i3, 0 ) ) || ( checkValence2Vertex( matrix, i2, 7 ) ) &&
        ( !( checkValence2Vertex( matrix, i1, 2 ) ) ) && ( !( checkValence2Vertex( matrix, i4, 5 ) ) ) )
    {
        SET_BIT( matrix[ i1 ], 2, 0 );
        SET_BIT( matrix[ i4 ], 5, 0 );
    }
    else
    {
        // Calcula tamanho partindo da aresta 2 de i1 e partindo da aresta 5 de i4. Depois soma

        //usleep(5000 * 1000);
        calcVal2PathSize( matrix, i1, 2, seg_size, width );

        calcVal2PathSize( matrix, i4, 5, seg_size, width );
        //cout << "i1 seg_size total: " << seg_size << endl;


        // Calcula tamanho partindo da aresta 0 de i3 e partindo de i2 da aresta 0. Depois soma

        //usleep(5000 * 1000);
        calcVal2PathSize( matrix, i3, 0, seg_size2, width );

        calcVal2PathSize( matrix, i2, 7, seg_size2, width );
        //cout << "  i3 seg_size total: " << seg_size2 << endl;

        // compara tamanho de cada curva e retira aresta da menor
        if( seg_size2 < seg_size )
        {
            //cout << "seg_size maior retira aresta 7 e 0" << endl;
            SET_BIT( matrix[ i2 ], 7, 0 );
            SET_BIT( matrix[ i3 ], 0, 0 );
        }
        else
        {
            //cout << "seg_size2 maior retira aresta 2 e 5" << endl;
            SET_BIT( matrix[ i1 ], 2, 0 );
            SET_BIT( matrix[ i4 ], 5, 0 );
        }
    }
    //}
}


__device__ char crossCheck_Window( char* graph, int width, int i, int j )
{
    const int WIN_SIZE = 49;
    const int WIN_WIDTH = 7;
    char graphWindow[ WIN_SIZE ];

    for( int j_ = 0; j_ < WIN_WIDTH; j_++ )
    {
        for( int i_ = 0; i_ < WIN_WIDTH; i_++ )
        {
            /* WIN_WIDTH x WIN_WIDTH window - offset (WIN_WIDTH-1)/2 to center i,j */
            graphWindow[ j_ * WIN_WIDTH +
                         i_ ] =
                graph[ ( ( j - ( WIN_WIDTH - 1 ) / 2 ) + j_ ) * width + ( ( i - ( WIN_WIDTH - 1 ) / 2 ) + i_ ) ];
        }
    }

    for( int j_ = 0; j_ < ( WIN_WIDTH - 1 ); j_++ )
    {
        for( int i_ = 0; i_ < ( WIN_WIDTH - 1 ); i_++ )
        {
            if( ( CHECK_BIT( graphWindow[ j_ * WIN_WIDTH + i_ ], 2 ) ) &&
                ( CHECK_BIT( graphWindow[ ( j_ + 1 ) * WIN_WIDTH + i_ ], 7 ) ) &&
                ( CHECK_BIT( graphWindow[ j_ * WIN_WIDTH + i_ + 1 ], 0 ) ) &&
                ( CHECK_BIT( graphWindow[ ( j_ + 1 ) * WIN_WIDTH + i_ + 1 ], 5 ) ) )
            {
                processHeuristicsWindow( graphWindow, i_, j_, WIN_WIDTH );
            }
        }
    }

    /* return center node */
    return graphWindow[ WIN_SIZE / 2 ];
}


__device__ void crossCheck_Heuristics( char* graph, char* graph_aux, int width, int height, int i, int j,
                                       bool* flagSync )
{
    //char nodeResult = graph_aux[ j * width + i ];


    /* Nodes indexes */
    int index = ( j * width + i );

    int i0 = ( ( j + 1 ) * width + i - 1 );
    int i1 = ( ( j + 1 ) * width + i );
    int i2 = ( ( j + 1 ) * width + i + 1 );
    int i3 = ( ( j ) * width + i - 1 );
    int i4 = ( ( j ) * width + i + 1 );
    int i5 = ( ( j - 1 ) * width + i - 1 );
    int i6 = ( ( j - 1 ) * width + i );
    int i7 = ( ( j - 1 ) * width + i + 1 );


    //   Nodes Structure
    //   +----+----+----+
    //   |    |    |    |
    //   | i0 | i1 | i2 |
    //   +----+----+----+
    //   |    | in |    |
    //   | i3 | dex| i4 |
    //   +----+----+----+
    //   |    |    |    |
    //   | i5 | i6 | i7 |
    //   +----+----+----+


//    #!
//    if ( (CHECK_BIT(graph_aux[i1],2)) &&
//         (CHECK_BIT(graph_aux[i2],7)) &&
//         (CHECK_BIT(graph_aux[i3],0)) &&
//         (CHECK_BIT(graph_aux[i4],5)) )
//    {
//        processHeuristics(graph, i, j, width, height);
//    }

    /* old order and graph aux use */

    if( ( CHECK_BIT( graph_aux[ i3 ], 2 ) ) &&
        ( CHECK_BIT( graph_aux[ i0 ], 7 ) ) &&
        ( CHECK_BIT( graph_aux[ index ], 0 ) ) &&
        ( CHECK_BIT( graph_aux[ i1 ], 5 ) ) )
    {
        if( !( processHeuristics2( graph_aux, i - 1, j, width, height ) ) )
        {
            SET_BIT( graph[ index ], 0, 0 );
        }
    }

    if( ( CHECK_BIT( graph_aux[ i5 ], 2 ) ) &&
        ( CHECK_BIT( graph_aux[ i3 ], 7 ) ) &&
        ( CHECK_BIT( graph_aux[ i6 ], 0 ) ) &&
        ( CHECK_BIT( graph_aux[ index ], 5 ) ) )
    {
        if( processHeuristics2( graph_aux, i - 1, j - 1, width, height ) )
        {
            SET_BIT( graph[ index ], 5, 0 );
        }
    }

    if( ( CHECK_BIT( graph_aux[ i6 ], 2 ) ) &&
        ( CHECK_BIT( graph_aux[ index ], 7 ) ) &&
        ( CHECK_BIT( graph_aux[ i7 ], 0 ) ) &&
        ( CHECK_BIT( graph_aux[ i4 ], 5 ) ) )
    {
        if( !( processHeuristics2( graph_aux, i, j - 1, width, height ) ) )
        {
            SET_BIT( graph[ index ], 7, 0 );
        }
    }

    if( ( CHECK_BIT( graph_aux[ index ], 2 ) ) &&
        ( CHECK_BIT( graph_aux[ i1 ], 7 ) ) &&
        ( CHECK_BIT( graph_aux[ i4 ], 0 ) ) &&
        ( CHECK_BIT( graph_aux[ i2 ], 5 ) ) )
    {
        if( processHeuristics2( graph_aux, i, j, width, height ) )
        {
            SET_BIT( graph[ index ], 2, 0 );
        }
    }

//    return;

    /* Scanline order (botton-top) */

//    if ( (CHECK_BIT(graph_aux[i5],2)) &&
//         (CHECK_BIT(graph_aux[i3],7)) &&
//         (CHECK_BIT(graph_aux[i6],0)) &&
//         (CHECK_BIT(graph_aux[index], 5)) )
//    {
//        if (check_neighboord(graph_aux, i-1, j-1, width))
//        {
//            nodeResult = crossCheck_Window(graph_aux, width, i, j);
//            graph[index] = nodeResult;
//            return;
//        }
//        else
//        {
//            if (processHeuristics2(graph_aux, i-1, j-1, width, height))
//               SET_BIT(graph[index], 5, 0);
//        }
//    }

//    if ( (CHECK_BIT(graph_aux[i6],2)) &&
//         (CHECK_BIT(graph_aux[index],7)) &&
//         (CHECK_BIT(graph_aux[i7],0)) &&
//         (CHECK_BIT(graph_aux[i4],5)) )
//    {
//        if (check_neighboord(graph_aux, i, j-1, width))
//        {
//            nodeResult = crossCheck_Window(graph_aux, width, i, j);
//            graph[index] = nodeResult;
//            return;
//        }
//        else
//        {
//            if (!processHeuristics2(graph_aux, i, j-1, width, height))
//               SET_BIT(graph[index], 7, 0);
//        }
//    }

//    if ( (CHECK_BIT(graph_aux[i3],2)) &&
//         (CHECK_BIT(graph_aux[i0],7)) &&
//         (CHECK_BIT(graph_aux[index],0)) &&
//         (CHECK_BIT(graph_aux[i1],5)) )
//    {
//        if (check_neighboord(graph_aux, i-1, j, width))
//        {
//            nodeResult = crossCheck_Window(graph_aux, width, i, j);
//            graph[index] = nodeResult;
//            return;
//        }
//        else
//        {
//            if (!processHeuristics2(graph_aux, i-1, j, width, height))
//               SET_BIT(graph[index], 0, 0);
//        }
//    }

//    if ( (CHECK_BIT(graph_aux[index],2)) &&
//         (CHECK_BIT(graph_aux[i1],7)) &&
//         (CHECK_BIT(graph_aux[i4],0)) &&
//         (CHECK_BIT(graph_aux[i2],5)) )
//    {
//        if (check_neighboord(graph_aux, i, j, width))
//        {
//            nodeResult = crossCheck_Window(graph_aux, width, i, j);
//            graph[index] = nodeResult;
//            return;
//        }
//        else
//        {
//            if (processHeuristics2(graph_aux, i, j, width, height))
//               SET_BIT(graph[index], 2, 0);
//        }
//    }

    //graph[index] = nodeResult;

//    if (processHeuristics2(graph_aux, x, y, width, height))
//        SET_BIT(graph[i1], 2, 0);

//    if (!(processHeuristics2(graph_aux, x, y+1, width, height)) )
//        SET_BIT(graph[i2], 7, 0);

//    if  (!(processHeuristics2(graph_aux, x+1, y, width, height)) )
//        SET_BIT(graph[i3], 5, 0);

//    if (processHeuristics2(graph_aux, x+1, y+1, width, height))
//        SET_BIT(graph[i4], 0, 0);


//    if (processHeuristics2(graph_aux, j-1, i-1, width, height))
//        SET_BIT(graph[i1], 5, 0);
//    if (processHeuristics2(graph_aux, j-1, i+1, width, height))
//        SET_BIT(graph[i1], 0, 0);
//    if (processHeuristics2(graph_aux, j+1, i-1, width, height))
//        SET_BIT(graph[i1], 2, 0);
//    if (processHeuristics2(graph_aux, j+1, i+1, width, height))
//        SET_BIT(graph[i1], 7, 0);

//    if (processHeuristics2(graph_aux, j-1, i, width, height))
//        SET_BIT(graph[i2], 5, 0);
//    if (processHeuristics2(graph_aux, j-1, i+2, width, height))
//        SET_BIT(graph[i2], 0, 0);
//    if (processHeuristics2(graph_aux, j+1, i, width, height))
//        SET_BIT(graph[i2], 2, 0);
//    if (processHeuristics2(graph_aux, j+1, i+2, width, height))
//        SET_BIT(graph[i2], 7, 0);

//    if (processHeuristics2(graph_aux, j, i-1, width, height))
//        SET_BIT(graph[i3], 5, 0);
//    if (processHeuristics2(graph_aux, j, i+1, width, height))
//        SET_BIT(graph[i3], 0, 0);
//    if (processHeuristics2(graph_aux, j+2, i-1, width, height))
//        SET_BIT(graph[i3], 2, 0);
//    if (processHeuristics2(graph_aux, j+2, i+1, width, height))
//        SET_BIT(graph[i3], 7, 0);

//    if (processHeuristics2(graph_aux, j, i, width, height))
//        SET_BIT(graph[i4], 5, 0);
//    if (processHeuristics2(graph_aux, j, i+2, width, height))
//        SET_BIT(graph[i4], 0, 0);
//    if (processHeuristics2(graph_aux, j+2, i, width, height))
//        SET_BIT(graph[i4], 2, 0);
//    if (processHeuristics2(graph_aux, j+2, i+2, width, height))
//        SET_BIT(graph[i4], 7, 0);
}


///* Change edges according to heuristics */
//__device__ bool crossCheck(char* graph, int width, int height, int i, int j){

//    for(int i = 0 ; i < height ; i++ ) {
//        for(int j = 0 ; j < width ; j++ ) {

//            /*    i2 --- i4
//                  |      |
//                  i1 --- i3    */

//            int i1 = (i*width + j);
//            int i2 = ((i+1)*width + j);
//            int i3 = (i*width + j+1);
//            int i4 = ((i+1)*width + j+1);

//            // Checa se blocos 2x2 estão totalmente conectados

//            /* Checa se blocos 2x2 estão totalmente conectados */

//            /*  3--- i2 ---(4)   (3)--- i4 ---4
//                    / | \              / | \
//                   /  |  \            /  |  \
//                  5  (6)  7          5  (6)  7


//                  0  (1)   2         0  (1)   2
//                   \  |   /           \  |   /
//                    \ |  /             \ |  /
//                 3--- i1 ---(4)   (3)--- i3 ---4         */

//            if ( (CHECK_BIT(graph[i1],1)) && (CHECK_BIT(graph[i1],4)) &&
//                 (CHECK_BIT(graph[i2],6)) && (CHECK_BIT(graph[i2],4)) &&
//                 (CHECK_BIT(graph[i3],3)) && (CHECK_BIT(graph[i3],1)) &&
//                 (CHECK_BIT(graph[i4],6)) && (CHECK_BIT(graph[i4],3)) )
//            {
//                SET_BIT(graph[i1],2,0);
//                SET_BIT(graph[i2],7,0);
//                SET_BIT(graph[i3],0,0);
//                SET_BIT(graph[i4],5,0);
//            } else {
//                return false;
//            }

//        }
//    }
//}

///* Change edges according to heuristics */
//__device__ void removeCross(char* graph, int width, int height, int i, int j){

////    for(int i = 0 ; i < height ; i++ ) {
////        for(int j = 0 ; j < width ; j++ ) {

//            /*    i2 --- i4
//                  |      |
//                  i1 --- i3    */

//            int i1 = (i*width + j);
//            int i2 = ((i+1)*width + j);
//            int i3 = (i*width + j+1);
//            int i4 = ((i+1)*width + j+1);

//            // Checa se blocos 2x2 estão totalmente conectados

//            /* Checa se blocos 2x2 estão totalmente conectados */

//            /*  3--- i2 ---(4)   (3)--- i4 ---4
//                    / | \              / | \
//                   /  |  \            /  |  \
//                  5  (6)  7          5  (6)  7


//                  0  (1)   2         0  (1)   2
//                   \  |   /           \  |   /
//                    \ |  /             \ |  /
//                 3--- i1 ---(4)   (3)--- i3 ---4         */

//            SET_BIT(graph[i1],2,0);
//            SET_BIT(graph[i2],7,0);
//            SET_BIT(graph[i3],0,0);
//            SET_BIT(graph[i4],5,0);

////        }
////    }
//}
