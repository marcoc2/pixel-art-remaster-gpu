#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <assert.h>
#include <helper_timer.h>

#include "point.cu"

#define CELL_SIZE 45

#include "graph_functions.cu"
#include "diagram_functions.cu"
#include "triangulate_functions.cu"
#include "subdivision_functions.cu"

//#define PIXEL( i, j, widthstep, n_channels ) ( ( j * ( widthstep ) ) + ( i * n_channels ) )

#define N_CHANNELS 3

#define CUDA_ERROR_CHECK

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError() __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char* file, const int line )
{
    #ifdef CUDA_ERROR_CHECK
    if( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
    #endif

    return;
}


inline void __cudaCheckError( const char* file, const int line )
{
    #ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }

    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
    #endif

    return;
}


/* Put pixel colors on an array */

__global__ void color_kernel( uchar4* colorPos,
                              unsigned int width, unsigned int height, char* img_data, int img_widthstep,
                              int* edge_count )
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;

    // calculate uv coordinates


    // calculate simple sine wave pattern
    if( ( i < width ) && ( j < height ) )
    {
        int node_index = ( ( j * width ) + i ) * CELL_SIZE;

        for( int t = 0; t < CELL_SIZE; t++ )
        {
//            if (t < (((edge_count[node_index]-2)*3)))
//            {

//                colorPos[node_index+t].x =  (unsigned char) img_data[PIXEL(i,j,img_widthstep, N_CHANNELS) + 2];
//                colorPos[node_index+t].y = (unsigned char) img_data[PIXEL(i,j,img_widthstep, N_CHANNELS) + 1];
//                colorPos[node_index+t].z =(unsigned char) img_data[PIXEL(i,j,img_widthstep, N_CHANNELS) + 0];
//                colorPos[node_index+t].w = (unsigned char)255;
//            } else
//            {
//                colorPos[node_index+t].x = (unsigned char)0;
//                colorPos[node_index+t].y = (unsigned char)128;
//                colorPos[node_index+t].z = (unsigned char)0;
//                colorPos[node_index+t].w = (unsigned char)255;
//            }
            colorPos[ node_index + t ].x = ( unsigned char ) img_data[ PIXEL( i, j, img_widthstep, N_CHANNELS ) + 2 ];
            colorPos[ node_index + t ].y = ( unsigned char ) img_data[ PIXEL( i, j, img_widthstep, N_CHANNELS ) + 1 ];
            colorPos[ node_index + t ].z = ( unsigned char ) img_data[ PIXEL( i, j, img_widthstep, N_CHANNELS ) + 0 ];
            colorPos[ node_index + t ].w = ( unsigned char )255;
        }
    }
}


/* Put vertex coordinates on an array for VBO */

template< typename T >
__global__ void position_kernel( float2* pos, T* diagram, int* edge_count, unsigned int width, unsigned int height )
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;

    // calculate uv coordinates


    // calculate simple sine wave pattern
    if( ( i < width ) && ( j < height ) )
    {
        int cell_index = ( ( j * width ) + i ) * CELL_SIZE;
        int node_index = ( ( j * width ) + i );
        for( int t = 0; t < CELL_SIZE; t++ )
        {
            if( t < ( ( ( edge_count[ node_index ] - 2 ) * 3 ) ) )
            {
                pos[ cell_index + t ].x = diagram[ cell_index + t ].x;
                pos[ cell_index + t ].y = diagram[ cell_index + t ].y;
            }
            else
            {
                pos[ cell_index + t ].x = -100.0f;
                pos[ cell_index + t ].y = -100.0f;
            }
        }
    }
}


__global__ void graph_Kernel( char* a, int size, int width, int height, int img_widthstep, char* graph_out )
{
    int i = ( blockIdx.x * blockDim.x + threadIdx.x );
    int j = ( blockIdx.y * blockDim.y + threadIdx.y );

    if( ( i < width ) && ( j < height ) )
    {
        for( int e = 0; e < 8; e++ )
        {
            if( diff( i, j, e, width, height, img_widthstep, a ) )
            {
                SET_BIT( graph_out[ ( j * width ) + i ], e, 0 );
            }
            else
            {
                SET_BIT( graph_out[ ( j * width ) + i ], e, 1 );
            }
        }
    }
}


__global__ void trivial_cross_Kernel( int width, int height, char* graph )
{
    int i = ( blockIdx.x * blockDim.x + threadIdx.x );
    int j = ( blockIdx.y * blockDim.y + threadIdx.y );

    if( ( ( i >= 0 ) && ( j >= 0 ) ) && ( ( i < width ) && ( j < height ) ) )
    {
        //if ((i==1) && (j ==1)){

        //crossCheck(graph, width, height, 0, 0);
        //crossCheck(graph, width, height);

        crossCheck_4( graph, width, height, i, j );
        //crossCheck_Heuristics(graph, width, height, i, j);
    }
}


__global__ void ambiguous_cross_Kernel( int width, int height, char* graph, char* graph_aux, bool* flagSync )
{
    int i = ( blockIdx.x * blockDim.x + threadIdx.x );
    int j = ( blockIdx.y * blockDim.y + threadIdx.y );

    if( ( i < width ) && ( j < height ) )
    {
        crossCheck_Heuristics( graph, graph_aux, width, height, i, j, flagSync );
    }
}


template< typename T >
__global__ void cells_Kernel( int width, int height, char* graph, T* diagram, int* edge_count )
{
    int i = ( blockIdx.x * blockDim.x + threadIdx.x );
    int j = ( blockIdx.y * blockDim.y + threadIdx.y );

    if( ( i < width ) && ( j < height ) )
    {
        int cell_index = ( ( j * width ) + i ) * CELL_SIZE;
        int node_index = ( ( j * width ) + i );


        edge_count[ ( j * width ) + i ] = createCellFromPattern< T >( graph[ ( j * width ) + i ],
                                                                      graph[ node_index - 1 ], graph[ node_index + 1 ],
                                                                      diagram, cell_index,
                                                                      i, j );


//        edge_count[(j*width) + i] = createCellFromPattern(64, 4, 0, diagram, cell_index,
//                              i, j);
    }
}


template< typename T >
__global__ void subdivision_Kernel( char* image_data, int width, int img_widthstep, int height, char* graph,
                                    T* diagram, T* diagram_aux, int* edge_count, int* edge_count_old, bool* edge_status,
                                    int* link_index )
{
    int i = ( blockIdx.x * blockDim.x + threadIdx.x );
    int j = ( blockIdx.y * blockDim.y + threadIdx.y );

    if( ( i < width ) && ( j < height ) )
    {
        int cell_index = ( ( j * width ) + i ) * CELL_SIZE;
        int node_index = ( ( j * width ) + i );
        //int old_e_count = edge_count[ node_index ];

        /* Do not process internal node */
        if( graph[ node_index ] == 90 )
        {
            return;
        }

        //checkEdges(&edge_status[cell_index], &diagram[ (node_index * CELL_SIZE)], graph[node_index], edge_count[node_index], i, j, &link_index[cell_index]);

        int new_edge_count = subdivision( image_data, &diagram[ cell_index ], diagram_aux, edge_count_old, node_index,
                                          width, img_widthstep, height, i, j, graph[ node_index ],
                                          &edge_status[ cell_index ],
                                          &link_index[ cell_index ] );

        __syncthreads();

        edge_count[ node_index ] = new_edge_count;

//        if ( (i == 2) && (j == 2) )
//        {
//            printf("edge status: ");
//            for (int t = 0; t < old_e_count; t++)
//                printf("%d, ", edge_status[cell_index + t]);
//            printf("\n");

//            printf("link index: ");
//            for (int t = 0; t < old_e_count; t++)
//                printf("%d, ", link_index[cell_index + t]);
//            printf("\n");
//            printf("node: %d\n", graph[node_index]);
//        }
    }
}


template< typename T >
__global__ void triangulate_Kernel( int width, int height, T* diagram, int* edge_count )
{
    int i = ( blockIdx.x * blockDim.x + threadIdx.x );
    int j = ( blockIdx.y * blockDim.y + threadIdx.y );


    if( ( i < width ) && ( j < height ) )
    {
        int cell_index = ( ( j * width ) + i ) * CELL_SIZE;
        int edge_c_index = ( ( j * width ) + i );

        /* Polygon Triangulation Function */
        triangulate_polygon( &diagram[ cell_index ], edge_count, edge_c_index, CELL_SIZE, i, j );

        /* Set the number of triangules for the polygon */
        //edge_count[edge_c_index] = (edge_count[edge_c_index] - 2);
    }
}


// Wrapper for the __global__ call that sets up the kernel call
extern "C" Point * launch_kernel( float2 * pos, uchar4 * colorPos, float time,
                                  char* img_data, int img_width, int img_height, int img_widthstep, int* edge_count_h,
                                  char* graph_h, bool subdivide )
{
    /* c_pattern.png */
    //char graph_dump[64] = {18, 26, 26, 25, 28, 26, 26, 10, 82, 90, 201, 17, 12, 116, 90, 74, 82, 202, 130, 18, 14, 32, 115, 74,
    //                       82, 74, 66, 82, 90, 186, 90, 74, 82, 74, 66, 82, 90, 93, 90, 74, 82, 78, 68, 80, 200, 1, 115, 74,
    //                       82, 90, 46, 48, 136, 147, 90, 74, 80, 88, 88, 56, 152, 88, 88, 72 };

    /* alex.png */
    //char graph_dump[576] = {(char) 18, (char) 26, (char) 26, (char) 26, (char) 10, (char) 18, (char) 25, (char) 24, (char) 24, (char) 28, (char) 26, (char) 26, (char) 25, (char) 24, (char) 24, (char) 28, (char) 10, (char) 18, (char) 26, (char) 26, (char) 26, (char) 26, (char) 26, (char) 10, (char) 82, (char) 90, (char) 90, (char) 90, (char) 78, (char) 196, (char) 20, (char) 26, (char) 26, (char) 10, (char) 114, (char) 202, (char) 18, (char) 26, (char) 26, (char) 9, (char) 97, (char) 83, (char) 90, (char) 90, (char) 90, (char) 90, (char) 90, (char) 74, (char) 82, (char) 90, (char) 90, (char) 90, (char) 90, (char) 46, (char) 36, (char) 112, (char) 88, (char) 72, (char) 82, (char) 74, (char) 80, (char) 88, (char) 200, (char) 130, (char) 146, (char) 90, (char) 90, (char) 90, (char) 90, (char) 90, (char) 90, (char) 74, (char) 82, (char) 90, (char) 90, (char) 90, (char) 90, (char) 90, (char) 42, (char) 34, (char) 18, (char) 10, (char) 82, (char) 74, (char) 18, (char) 26, (char) 10, (char) 66, (char) 82, (char) 90, (char) 90, (char) 90, (char) 90, (char) 90, (char) 90, (char) 74, (char) 82, (char) 90, (char) 90, (char) 90, (char) 89, (char) 92, (char) 74, (char) 66, (char) 82, (char) 74, (char) 82, (char) 73, (char) 83, (char) 90, (char) 74, (char) 70, (char) 80, (char) 92, (char) 90, (char) 90, (char) 90, (char) 90, (char) 90, (char) 74, (char) 82, (char) 90, (char) 90, (char) 201, (char) 17, (char) 12, (char) 96, (char) 67, (char) 82, (char) 78, (char) 192, (char) 147, (char) 90, (char) 90, (char) 73, (char) 81, (char) 56, (char) 12, (char) 116, (char) 90, (char) 90, (char) 90, (char) 90, (char) 74, (char) 82, (char) 90, (char) 202, (char) 130, (char) 16, (char) 8, (char) 180, (char) 74, (char) 82, (char) 90, (char) 186, (char) 90, (char) 90, (char) 202, (char) 130, (char) 16, (char) 24, (char) 8, (char) 34, (char) 114, (char) 90, (char) 90, (char) 90, (char) 74, (char) 82, (char) 90, (char) 78, (char) 68, (char) 20, (char) 26, (char) 9, (char) 98, (char) 82, (char) 90, (char) 90, (char) 90, (char) 90, (char) 74, (char) 66, (char) 16, (char) 28, (char) 10, (char) 66, (char) 82, (char) 90, (char) 90, (char) 90, (char) 74, (char) 82, (char) 90, (char) 90, (char) 46, (char) 36, (char) 228, (char) 4, (char) 64, (char) 83, (char) 89, (char) 92, (char) 90, (char) 90, (char) 73, (char) 65, (char) 19, (char) 9, (char) 97, (char) 65, (char) 83, (char) 90, (char) 90, (char) 90, (char) 74, (char) 82, (char) 90, (char) 90, (char) 90, (char) 46, (char) 36, (char) 32, (char) 178, (char) 202, (char) 18, (char) 10, (char) 114, (char) 206, (char) 128, (char) 147, (char) 202, (char) 130, (char) 130, (char) 146, (char) 90, (char) 90, (char) 90, (char) 90, (char) 74, (char) 82, (char) 90, (char) 90, (char) 90, (char) 90, (char) 42, (char) 38, (char) 80, (char) 72, (char) 80, (char) 72, (char) 80, (char) 92, (char) 186, (char) 89, (char) 72, (char) 65, (char) 65, (char) 83, (char) 90, (char) 90, (char) 90, (char) 90, (char) 74, (char) 82, (char) 90, (char) 90, (char) 90, (char) 90, (char) 73, (char) 81, (char) 56, (char) 24, (char) 24, (char) 24, (char) 24, (char) 8, (char) 224, (char) 16, (char) 136, (char) 134, (char) 148, (char) 90, (char) 90, (char) 90, (char) 90, (char) 90, (char) 74, (char) 82, (char) 90, (char) 90, (char) 90, (char) 201, (char) 129, (char) 19, (char) 25, (char) 28, (char) 26, (char) 26, (char) 26, (char) 14, (char) 20, (char) 26, (char) 14, (char) 80, (char) 44, (char) 116, (char) 90, (char) 90, (char) 90, (char) 90, (char) 74, (char) 82, (char) 90, (char) 90, (char) 202, (char) 130, (char) 146, (char) 201, (char) 16, (char) 8, (char) 115, (char) 89, (char) 88, (char) 92, (char) 42, (char) 114, (char) 93, (char) 58, (char) 10, (char) 34, (char) 114, (char) 90, (char) 90, (char) 90, (char) 74, (char) 82, (char) 90, (char) 90, (char) 78, (char) 68, (char) 192, (char) 20, (char) 10, (char) 146, (char) 202, (char) 20, (char) 26, (char) 10, (char) 98, (char) 194, (char) 4, (char) 116, (char) 74, (char) 66, (char) 82, (char) 90, (char) 90, (char) 90, (char) 74, (char) 82, (char) 90, (char) 90, (char) 90, (char) 42, (char) 50, (char) 10, (char) 98, (char) 82, (char) 74, (char) 2, (char) 114, (char) 74, (char) 66, (char) 66, (char) 2, (char) 34, (char) 98, (char) 66, (char) 82, (char) 90, (char) 90, (char) 90, (char) 74, (char) 82, (char) 90, (char) 90, (char) 90, (char) 73, (char) 81, (char) 72, (char) 66, (char) 82, (char) 74, (char) 64, (char) 83, (char) 73, (char) 65, (char) 67, (char) 66, (char) 66, (char) 66, (char) 66, (char) 82, (char) 90, (char) 90, (char) 90, (char) 74, (char) 82, (char) 90, (char) 90, (char) 202, (char) 130, (char) 6, (char) 4, (char) 64, (char) 83, (char) 78, (char) 144, (char) 200, (char) 129, (char) 147, (char) 73, (char) 66, (char) 66, (char) 66, (char) 66, (char) 82, (char) 90, (char) 90, (char) 90, (char) 74, (char) 82, (char) 90, (char) 90, (char) 78, (char) 68, (char) 84, (char) 46, (char) 180, (char) 90, (char) 90, (char) 57, (char) 136, (char) 147, (char) 206, (char) 4, (char) 64, (char) 67, (char) 64, (char) 67, (char) 82, (char) 90, (char) 90, (char) 90, (char) 74, (char) 82, (char) 90, (char) 90, (char) 90, (char) 42, (char) 34, (char) 114, (char) 46, (char) 112, (char) 200, (char) 19, (char) 154, (char) 90, (char) 90, (char) 46, (char) 176, (char) 72, (char) 146, (char) 73, (char) 83, (char) 90, (char) 90, (char) 90, (char) 74, (char) 82, (char) 90, (char) 90, (char) 90, (char) 78, (char) 68, (char) 84, (char) 90, (char) 58, (char) 154, (char) 90, (char) 90, (char) 90, (char) 90, (char) 90, (char) 58, (char) 10, (char) 194, (char) 146, (char) 90, (char) 90, (char) 90, (char) 90, (char) 74, (char) 82, (char) 90, (char) 90, (char) 90, (char) 90, (char) 46, (char) 36, (char) 112, (char) 92, (char) 90, (char) 90, (char) 90, (char) 90, (char) 90, (char) 90, (char) 89, (char) 72, (char) 65, (char) 83, (char) 90, (char) 90, (char) 90, (char) 90, (char) 74, (char) 82, (char) 90, (char) 90, (char) 90, (char) 90, (char) 90, (char) 46, (char) 48, (char) 12, (char) 112, (char) 88, (char) 88, (char) 88, (char) 88, (char) 200, (char) 17, (char) 136, (char) 147, (char) 90, (char) 90, (char) 90, (char) 90, (char) 90, (char) 74, (char) 80, (char) 88, (char) 88, (char) 88, (char) 88, (char) 88, (char) 88, (char) 56, (char) 8, (char) 48, (char) 24, (char) 24, (char) 24, (char) 24, (char) 136, (char) 16, (char) 152, (char) 88, (char) 88, (char) 88, (char) 88, (char) 88, (char) 88, (char) 72 };

    StopWatchInterface* timer = NULL;
    sdkCreateTimer( &timer );
    sdkResetTimer( &timer );
    sdkStartTimer( &timer );

    /* Size needed to hold image data */
    size_t img_size = img_widthstep * img_height * sizeof( char );


    //size_t size = N_CHANNELS * img_width * img_height * sizeof( char );


    /*----------- Image Device -------------*/

    /* Image data pointer to device memory */
    char* img_data_d;

    /* Allocates memory in device to image */
    hipMalloc( ( void** ) &img_data_d, img_size );

    /* Copy data from host to device */
    hipMemcpy( img_data_d, img_data, img_size, hipMemcpyHostToDevice );


    /*----------- Graph Device -------------*/

    /* Graph data pointer to device memory */
    char* graph_d;

    /* Allocates memory in device to graph */
    hipMalloc( ( void** ) &graph_d, img_width * img_height * sizeof( char ) );

    /* Aux Graph data pointer to device memory */
    char* graph_d_aux;

    /* Allocates memory in device to graph */
    hipMalloc( ( void** ) &graph_d_aux, img_width * img_height * sizeof( char ) );

    /* ------- TEST FOR NOT FERMI GPUS  -------- */
    /* Copy data from host to device */
    hipMemcpy( graph_d, graph_h, img_width * img_height * sizeof( char ), hipMemcpyHostToDevice );


    /*----------- Diagram Device -------------*/

    /* Diagram data pointer to host memory */
    Point* diagram_h;
    /* Diagram data pointer to device memory */
    Point* diagram_d;
    /* Auxiliar diagram data pointer to device memory (to be used on subdivision) */
    Point* diagram_aux_d;

    /* Allocates memory in host to diagram */
    diagram_h = ( Point* )malloc( img_width * img_height * sizeof( Point ) * CELL_SIZE );
    /* Allocates memory in device to diagram */
    hipMalloc( ( void** ) &diagram_d, img_width * img_height * sizeof( Point ) * CELL_SIZE );
    /* Allocates memory in device to auxiliar diagram */
    hipMalloc( ( void** ) &diagram_aux_d, img_width * img_height * sizeof( Point ) * CELL_SIZE );


    /*----------- Polygon Edge Count Device -------------*/

    /* Int array data pointer to device memory */
    //int* edge_count_h;
    /* Int array data pointer to host memory */
    int* edge_count_d;
    /* Auxiliar Int array data pointer to host memory */
    int* edge_count_aux_d;

    /* Allocates memory in host to array of vertices count */
    //edge_count_h = (int*)malloc(img_width*img_height*sizeof(int));
    /* Allocates memory in device to array of vertices count */
    hipMalloc( ( void** ) &edge_count_d, img_width * img_height * sizeof( int ) );
    /* Allocates memory in device to array of vertices count */
    hipMalloc( ( void** ) &edge_count_aux_d, img_width * img_height * sizeof( int ) );

    /*-------------- Flag Array to Synchronize (still unused) ---------------*/

    bool* flagSync_d;
    hipMalloc( ( void** ) &flagSync_d, img_width * img_height * sizeof( bool ) );


    /*-------------- Arrays needed on Subdivision Kernel ---------------*/

    bool* edge_status_d;

    hipMalloc( ( void** ) &edge_status_d, img_width * img_height * sizeof( bool ) * CELL_SIZE );

    int* link_index_d;

    hipMalloc( ( void** ) &link_index_d, img_width * img_height * sizeof( int ) * CELL_SIZE );


    /* set grid and blocks */
    dim3 threadsPerBlock( 2, 2 );
    dim3 numBlocks( ( img_width / threadsPerBlock.x ) + ( img_width % threadsPerBlock.x == 0 ? 0 : 1 ),
                    ( img_height / threadsPerBlock.y ) + ( img_height % threadsPerBlock.y == 0 ? 0 : 1 ) );

    printf( "numblocks.x = %d\n", numBlocks.x );
    printf( "numblocks.y = %d\n", numBlocks.y );

    /* ********************* Beginning of the Kernel Pipeline ********************* */

    /* Graph Stage */
    graph_Kernel <<< numBlocks, threadsPerBlock >>> ( img_data_d,
                                                      img_size,
                                                      img_width,
                                                      img_height,
                                                      img_widthstep,
                                                      graph_d );
    hipDeviceSynchronize();

    /* Graph's Cross Check Stage (trivial case) */
    trivial_cross_Kernel <<< numBlocks, threadsPerBlock >>> ( img_width,
                                                              img_height,
                                                              graph_d );

    hipMemcpy( graph_d_aux, graph_d, img_width * img_height * sizeof( char ), hipMemcpyDeviceToDevice );
    hipDeviceSynchronize();

    /* Graph's Heuristics Stage */
    ambiguous_cross_Kernel <<< numBlocks, threadsPerBlock >>> ( img_width,
                                                                img_height,
                                                                graph_d,
                                                                graph_d_aux,
                                                                flagSync_d );

    /* Diagram Stage */
    cells_Kernel< Point > <<< numBlocks, threadsPerBlock >>> ( img_width,
                                                               img_height,
                                                               graph_d,
                                                               diagram_d,
                                                               edge_count_d );

    //cells_Kernel<float2> <<< numBlocks, threadsPerBlock >>> (img_width, img_height, graph_d, pos, edge_count_d);

    hipMemcpy( diagram_aux_d, diagram_d,
                img_width * img_height * sizeof( Point ) * CELL_SIZE, hipMemcpyDeviceToDevice );
    hipMemcpy( edge_count_aux_d, edge_count_d,
                img_width * img_height * sizeof( int ), hipMemcpyDeviceToDevice );

    /* Cell Smoothing Stage */
    if( subdivide )
    {
        subdivision_Kernel< Point > <<< numBlocks, threadsPerBlock >>> ( img_data_d,
                                                                         img_width,
                                                                         img_widthstep,
                                                                         img_height,
                                                                         graph_d,
                                                                         diagram_d,
                                                                         diagram_aux_d,
                                                                         edge_count_d,
                                                                         edge_count_aux_d,
                                                                         edge_status_d,
                                                                         link_index_d );
    }

    /* Cell's Triangulation Stage */
    triangulate_Kernel< Point > <<< numBlocks, threadsPerBlock >>> ( img_width,
                                                                     img_height,
                                                                     diagram_d,
                                                                     edge_count_d );
    //triangulate_cell<float2> <<< numBlocks, threadsPerBlock >>> (img_width, img_height, pos, edge_count_d, CELL_SIZE);

    /* Put pixel colors on an array */
    color_kernel <<< numBlocks, threadsPerBlock >>> ( colorPos,
                                                      img_width,
                                                      img_height,
                                                      img_data_d,
                                                      img_widthstep,
                                                      edge_count_d );

    /* Put vertex coordinates on an array for VBO */
    position_kernel< Point ><<< numBlocks, threadsPerBlock >>> ( pos,
                                                                 diagram_d,
                                                                 edge_count_d,
                                                                 img_width,
                                                                 img_height );

    /* Check for errors */
    CudaCheckError();

    /* ********************* End of the Kernel Pipeline ********************* */

    sdkStopTimer( &timer );
    float kernelTime = sdkGetTimerValue( &timer );
    sdkDeleteTimer( &timer );

    printf( "Time for the kernels: %f ms\n", kernelTime );

    hipMemcpy( edge_count_h, edge_count_d,
                img_width * img_height * sizeof( int ), hipMemcpyDeviceToHost );
    hipMemcpy( diagram_h, diagram_d,
                img_width * img_height * sizeof( Point ) * CELL_SIZE, hipMemcpyDeviceToHost );
    hipMemcpy( graph_h, graph_d,
                img_width * img_height * sizeof( char ), hipMemcpyDeviceToHost );

    //pos = (float2*)diagram_h;

    /* Degub pixel/node/cell i x j */

    //    int i = 8;
    //    int j = 5;
    //    int index = (i*img_width) + j;
    //    int index_cell = ( (i*img_width) + j ) * CELL_SIZE;

    //    printf("edge_count[i]: %d\n", edge_count_h[index]);
    //    printf("node[i]: %d\n", (unsigned int) (unsigned char) graph_h[index]);

    //    printf("cell_index: %d :\n", index_cell);
    //    for (int t = 0; t < CELL_SIZE; t++){
    //        //cout << "P( " << diagram_h[i].x << ", " << diagram_h[i].y << " )" << endl;
    //        printf("P( %2.2f, %2.2f )\n", diagram_h[index_cell + t].x, diagram_h[index_cell + t].y);
    //    }

    /* clean up */
    hipFree( img_data_d );
    hipFree( graph_d );
    hipFree( graph_d_aux );
    hipFree( diagram_d );
    hipFree( diagram_aux_d );
    hipFree( flagSync_d );
    hipFree( edge_status_d );
    hipFree( edge_count_d );
    hipFree( edge_count_aux_d );
    hipFree( link_index_d );

    return diagram_h;
}
