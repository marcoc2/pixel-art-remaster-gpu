
#include <hip/hip_runtime.h>
//#include <cuda.h>

/** Check bit value */
#define CHECK_BIT(var,pos) ((var) & (1<<(pos)))

/** Clean bit then set */
#define SET_BIT(var,pos,data) var &= ~(1 << pos); var |= data << pos

#define Ymask 0x00FF0000
#define Umask 0x0000FF00
#define Vmask 0x000000FF
/* Original trY  */
#define trY   0x00300000
/* Better value when not using Splines with difusion (Empirical value) */
//#define trY   0x00050000
#define trU   0x00000700
#define trV   0x00000006

#define RGBA  0x00000000

#define PIXEL(i, j, width, n_channels) ((j*(n_channels*width))+(i*n_channels))

// Retorna posição do bit da aresta da célula vizinha
__device__ int conected_edge(int edge)
{
    return (8-1)-edge;

    /*    switch(edge)
        {
            case 0: return 7;
            case 1: return 6;
            case 2: return 5;
            case 3: return 4;
            case 4: return 3;
            case 5: return 2;
            case 6: return 1;
            case 7: return 0;
        }
    */
}


// Calcula índice da célula vizinha de acordo com a aresta conectada
__device__ int calc_index(int index, int edge, int width)
{
    //int result;
    switch(edge)
    {
    case 0: return index + width - 1;
    case 1: return index + width;
    case 2: return index + width + 1;
    case 3: return index - 1;
    case 4: return index + 1;
    case 5: return index - width - 1;
    case 6: return index - width;
    case 7: return index - width + 1;
    }
    //return result;
}

/* Initalize RGB to YUV lookup table */
__device__ unsigned int RGBtoYUV(int c)
{
    int r, g, b, y, u, v;

    r = (c & 0x0000FF);
    g = (c & 0x00FF00) >> 8;
    b = (c & 0xFF0000) >> 16 ;
    //r = (c & 0xFF0000) >> 16;
    //g = (c & 0x00FF00) >> 8;
    //b = c & 0x0000FF;
    y = int((0.299*float(r) + 0.587*float(g) + 0.114*float(b)));
    //u = (unsigned int)(-0.169*r - 0.331*g + 0.5*b) + 128; //Cb
    //v = (unsigned int)(0.5*r - 0.419*g - 0.081*b) + 128;  //Cr
    u = int(((b - y) * 0.492f));
    v = int(((r - y) * 0.877f));


    return (unsigned int)((y << 16) + (u << 8) + v);

}

/* Convert value to int type */
__device__ unsigned int DATAtoINT(unsigned int r,unsigned int g,unsigned int b){

    unsigned int result;

    /* R_ = xxxx xxxx xxxx xxxx xxxx xxxx RRRR RRRR*/
    unsigned int R_ = (unsigned int) r;

    /* G_ = xxxx xxxx xxxx xxxx GGGG GGGG xxxx xxxx*/
    unsigned int G_ = (unsigned int) g << 8;

    /* B_ = xxxx xxxx BBBB BBBB xxxx xxxx rrrr rrrr*/
    unsigned int B_ = (unsigned int) b << 16;
    //unsigned int A_ = 0x00FFFFFF;


    /* R_ = 0000 0000 0000 0000 0000 0000 RRRR RRRR*/
    R_ &= 0x000000FF;

    /* G_ = 0000 0000 0000 0000 GGGG GGGG 0000 0000*/
    G_ = (G_ & 0x0000FF00);

    /* B_ = 0000 0000 BBBB BBBB 0000 0000 0000 0000*/
    B_ = (B_ & 0x00FF0000);


    /* result = 0000 0000 BBBB BBBB GGGG GGGG RRRR RRRR*/
    result = ((R_|G_|B_));

    return result;
}

int __device__ abs_(int a){
    if (a < 0)
        return (-1 * a);
}

__device__ int diff(int w,int h, int pos, int width, int height, char* pixel_data){
    int result;
    unsigned int   YUV1;
    unsigned int   YUV2;

    /* Pixels colors in RGB to be converted */
    char* pixel_src, *pixel_dst;

    /* Pixels colors in YUV space to be compared */
    unsigned int pixel_src_YUV, pixel_dst_YUV;

    /* Height and Width position of pixel_dst */
    int n_h, n_w;

    //   +----+----+----+
    //   |    |    |    |
    //   | 0  | 1  | 2  |
    //   +----+----+----+
    //   |    |    |    |
    //   | 3  | x  | 4  |
    //   +----+----+----+
    //   |    |    |    |
    //   | 5  | 6  | 7  |
    //   +----+----+----+

    //    uchar r = img->imageData[h*img->widthStep + w*nChannels];
    //    uchar g = img->imageData[h*img->widthStep + w*nChannels+1];
    //    uchar b = img->imageData[h*img->widthStep + w*nChannels+2];

    pixel_src = &(pixel_data[PIXEL(w, h, width, 3)]);

    // __DEBUG__
//    printf("w: %d, h: %d, pos: %d\n", w, h, pos);
//    printf("pixel_src[0]: %d\n", (unsigned char) pixel_src[0]);
//    printf("pixel_src[1]: %d\n", (unsigned char) pixel_src[1]);
//    printf("pixel_src[2]: %d\n", (unsigned char) pixel_src[2]);

    pixel_src_YUV = DATAtoINT(pixel_src[0], pixel_src[1], pixel_src[2]);

    switch (pos){
    case 0:
        n_w = (w-1); n_h = (h+1);
        break;
    case 1:
        n_w = (w); n_h = (h+1);
        break;
    case 2:
        n_w = (w+1); n_h = (h+1);
        break;
    case 3:
        n_w = (w-1); n_h = (h);
        break;
    case 4:
        n_w = (w+1); n_h = (h);
        break;
    case 5:
        n_w = (w-1); n_h = (h-1);
        break;
    case 6:
        n_w = (w); n_h = (h-1);
        break;
    case 7:
        n_w = (w+1); n_h = (h-1);
        break;
    }


    //cuPrintf("pos: %d, n_w: %d, n_h: %d\n", pos, n_w, n_h);


    //index = n_h*img->getWidth() + n_w;

    /* Check if index is inside image boundaries */
    //if ((index > 0) && (index < ((img->getWidth())*(img->getHeight())-1)))
    if (!((n_w < 0) || (n_h < 0) ||
            (n_w >= width) ||
            (n_h >= height) ))
    {
        pixel_dst = &(pixel_data[PIXEL(n_w, n_h, width, 3)]);

        pixel_dst_YUV = DATAtoINT(pixel_dst[0], pixel_dst[1], pixel_dst[2]);

        if ((w == 1) && (h==1)){
            //cuPrintf("pos: %d, pixel_dst[0]: %d, pixel_dst[1]: %d, pixel_dst[2]: %d\n",
            //         pos, (unsigned char) pixel_dst[0], (unsigned char) pixel_dst[1], (unsigned char) pixel_dst[2]);
        }


        /* Mask against RGB_MASK to discard the alpha channel */
        YUV1 = RGBtoYUV(pixel_src_YUV);
        YUV2 = RGBtoYUV(pixel_dst_YUV);

        if ((w == 1) && (h==1)){
//            cuPrintf("pos: %d, YUV_src: %d, YUV_dest: %d, pixel_dst_YUV[0]: %d   , pixel_dst_YUV[1]: %d   , pixel_dst_YUV[2]: %d\n",
//                     pos,
//                     (YUV1 & Ymask),
//                     (YUV2 & Ymask),
//                     ( ( YUV1 & Ymask ) - ( YUV2 & Ymask ) ),
//                     abs_( ( YUV1 & Umask ) - ( YUV2 & Umask ) ),
//                     abs_( ( YUV1 & Vmask ) - ( YUV2 & Vmask ) ));
        }

        result = ( ( fabsf((YUV1 & Ymask) - (YUV2 & Ymask)) > trY ) ||
                   ( fabsf((YUV1 & Umask) - (YUV2 & Umask)) > trU ) ||
                   ( fabsf((YUV1 & Vmask) - (YUV2 & Vmask)) > trV ) );

        /* Print function data */

        //std::cout << "index src - x: " << w << " y:  " << h << std::endl;
        //std::cout << "index dst - x: " << n_w << " y:  " << n_h << std::endl;
        //std::cout << "pos: " << pos << " - ";
        //std::cout << "results: " << result << std::endl;

        //if (result != 0) {cout << result << endl;}
        return result;
    } else {return 1;}
}

__device__ bool checkValence2Edge(char* graph, int index_1, int index_2, int edge_1, int edge_2){
    int k_n;     // last conected edge
    int sum_1 = 0;
    int sum_2 = 0; // soma de arestas conectadas além da 2
    for (int k = (edge_1 + 1); k < (edge_1 + 8); k++)
    {
        if (CHECK_BIT(graph[index_1],k%8))
        {
            sum_1++;
            k_n = k;
        }
    }
    for (int k = (edge_2 + 1); k < (edge_2 + 8); k++)
    {
        if (CHECK_BIT(graph[index_2],k%8))
        {
            sum_2++;
            k_n = k;
        }
    }
    //cout << "sum_1: " << sum_1 << endl;
    //cout << "sum_2: " << sum_2 << endl;
    if ((sum_1 == 1) && (sum_2 == 1)) return true;
    else return false;
}

/* Varre o nó para saber se existe algum outro bit ativo
   Retorna true se apenas edge_1 está ativo             */
__device__ bool checkValence2Vertex(char* graph, int index_1, int edge_1){
    int k_n;     // last conected edge
    int sum = 0; // soma de arestas conectadas além da 2
    for (int k = (edge_1 + 1); k < (edge_1 + 8); k++)
    {
        if (CHECK_BIT(graph[index_1],k%8))
        {
            sum++;
            k_n = k;
        }
    }
    //cout << sum << endl;
    if (sum == 0) return true;
    else return false;
}

/* Calculate the size of a valence-2 path starting prom index */
__device__ int calcVal2PathSize(char* graph, int index, int edge, int result, int width)
{
    int k_n; // Outra aresta quando o segmento é de valência 2
    int sum = 0;
    //switch (side){
    //    case 0: {
            // a partir de edge + 1 (edge já é conexo)
             for (int k = (edge + 1); k < (edge + 7); k++)
             {
                 if (CHECK_BIT(graph[index],k%8))
                 {
                     sum++;
                     k_n = k;
                 }
             }
             // Compare to a big number to not enter in loop
             if (result > 30) return result;
             if (sum == 1)
             {
                 result++;
                 calcVal2PathSize(graph, calc_index(index, k_n%8, width), conected_edge(k_n%8), result, width);
             } else if (sum == 0) { return result;}
             return result;
        //}
    //}
}

__device__ void processHeuristics(char* graph, int j, int i, int width, int height){
    int seg_size = 0;
    int seg_size2 = 0;

    int i1 = (i*width + j);
    int i2 = ((i+1)*width + j);
    int i3 = (i*width + j+1);
    int i4 = ((i+1)*width + j+1);

    // Both edges are
    if ((checkValence2Edge(graph, i1, i4, 2, 5)) && (checkValence2Edge(graph, i3, i2, 0, 7)))
    {

    } else {
        /* Just i1-i4 edge is valence 2 */
        /*  i2  /i4
               /
            i1/  i3  */
        if (checkValence2Edge(graph, i1, i4, 2, 5)){
            /* Remove i2-i3 edge */
            SET_BIT(graph[i2],7,0);
            SET_BIT(graph[i3],0,0);
        } else
            /* Just i2-i3 edge is valence 2 */
            /*  i2\  i4
                   \
                i1  \i3  */
            if(checkValence2Edge(graph, i3, i2, 0, 7)){
                /* Remove i1-i4 edge */
                SET_BIT(graph[i1],2,0);
                SET_BIT(graph[i4],5,0);
            } else
                if( ( (checkValence2Vertex(graph, i1, 2)) || (checkValence2Vertex(graph, i4, 5)) ) &&
                        (!(checkValence2Vertex(graph, i3, 0))) && (!(checkValence2Vertex(graph, i2, 7))) ){
                    SET_BIT(graph[i2],7,0);
                    SET_BIT(graph[i3],0,0);
                } else
                    if( (checkValence2Vertex(graph, i3, 0)) || (checkValence2Vertex(graph, i2, 7)) &&
                            (!(checkValence2Vertex(graph, i1, 2))) && (!(checkValence2Vertex(graph, i4, 5))) ){
                        SET_BIT(graph[i1],2,0);
                        SET_BIT(graph[i4],5,0);
                    } else {

            // Calcula tamanho partindo da aresta 2 de i1 e partindo da aresta 5 de i4. Depois soma

            //usleep(5000 * 1000);
            seg_size = calcVal2PathSize(graph, i1,2, 0, width);

            seg_size = seg_size + calcVal2PathSize(graph,i4, 5, 0, width);
            //cout << "i1 seg_size total: " << seg_size << endl;


            // Calcula tamanho partindo da aresta 0 de i3 e partindo de i2 da aresta 0. Depois soma

            //usleep(5000 * 1000);
            seg_size2 = calcVal2PathSize(graph,i3,0, 0, width);

            seg_size2 = seg_size2 + calcVal2PathSize(graph,i2, 7, 0, width);
            //cout << "  i3 seg_size total: " << seg_size2 << endl;

            // compara tamanho de cada curva e retira aresta da menor
            if (seg_size2 < seg_size)
            {
                //cout << "seg_size maior retira aresta 7 e 0" << endl;
                SET_BIT(graph[i2],7,0);
                SET_BIT(graph[i3],0,0);
            } else
            {
                //cout << "seg_size2 maior retira aresta 2 e 5" << endl;
                SET_BIT(graph[i1],2,0);
                SET_BIT(graph[i4],5,0);
            }
        }
    }
}


/* Change edges according to heuristics */
__device__ void crossCheck(char* graph, int width, int height){

    for(int i = 0 ; i < height ; i++ ) {
        for(int j = 0 ; j < width ; j++ ) {

            /*    i2 --- i4
                  |      |
                  i1 --- i3    */

            int i1 = (i*width + j);
            int i2 = ((i+1)*width + j);
            int i3 = (i*width + j+1);
            int i4 = ((i+1)*width + j+1);

            // Checa se blocos 2x2 estão totalmente conectados

            /* Checa se blocos 2x2 estão totalmente conectados */

            /*  3--- i2 ---(4)   (3)--- i4 ---4
                    / | \              / | \
                   /  |  \            /  |  \
                  5  (6)  7          5  (6)  7


                  0  (1)   2         0  (1)   2
                   \  |   /           \  |   /
                    \ |  /             \ |  /
                 3--- i1 ---(4)   (3)--- i3 ---4         */

            if ( (CHECK_BIT(graph[i1],1)) && (CHECK_BIT(graph[i1],4)) &&
                 (CHECK_BIT(graph[i2],6)) && (CHECK_BIT(graph[i2],4)) &&
                 (CHECK_BIT(graph[i3],3)) && (CHECK_BIT(graph[i3],1)) &&
                 (CHECK_BIT(graph[i4],6)) && (CHECK_BIT(graph[i4],3)) )
            {
                //cvCircle( graph_img, cvPoint(n_j+half_sg,n_i+half_sg), 3, CV_RGB( 0,  0, 0 ), CV_FILLED, 8, 0 );
                //remove arestas cruzadas
                SET_BIT(graph[i1],2,0);
                SET_BIT(graph[i2],7,0);
                SET_BIT(graph[i3],0,0);
                SET_BIT(graph[i4],5,0);
            }
            // Checa pelas arestas cruzadas

            /*  3--- i2 ---4        3--- i4 ---4
                    / | \              / | \
                   /  |  \            /  |  \
                  5   6   (7)       (5)  6   7


                  0   1   (2)       (0)  1    2
                   \  |   /           \  |   /
                    \ |  /             \ |  /
                 3--- i1 ---4       3--- i3 ---4         */

            if ( (CHECK_BIT(graph[i1],2)) &&
                 (CHECK_BIT(graph[i2],7)) &&
                 (CHECK_BIT(graph[i3],0)) &&
                 (CHECK_BIT(graph[i4],5)) )
            {
                processHeuristics(graph, j, i, width, height);
            }
        }
    }
}
