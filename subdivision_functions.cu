#include "hip/hip_runtime.h"
/* Functions considering counter clockwise order for cells points */

__device__ inline int mod( int i, int n )
{
    return ( n + ( i % n ) ) % n;
}


__device__ int c_neighbor_index( int index, int edge, int width )
{
    //int result;
    switch( edge )
    {
        case 0:
            return index + width - 1;

        case 1:
            return index + width;

        case 2:
            return index + width + 1;

        case 3:
            return index - 1;

        case 4:
            return index + 1;

        case 5:
            return index - width - 1;

        case 6:
            return index - width;

        case 7:
            return index - width + 1;
    }
    return 0;
}


template< typename T >
__device__ T getQ_i( int i, T* diagram, int edge_count, char node )
{
    /* Nodes exception:
    36  -> diagonal "/"
    129 -> diagonal "\"
    These non border edges nodes are adjacent to TWO cells   */

    T P;
    int p_index = i % edge_count;
    if( p_index < 0 )
    {
        p_index = p_index + edge_count;
    }
    P.x = diagram[ p_index ].x;
    P.y = diagram[ p_index ].y;

    T P_next;
    P_next.x = diagram[ ( i + 1 ) % edge_count ].x;
    P_next.y = diagram[ ( i + 1 ) % edge_count ].y;

    T Qi;

    float lenght = sqrt( ( ( P_next.x - P.x ) * ( P_next.x - P.x ) + ( P_next.y - P.y ) * ( P_next.y - P.y ) ) );

    //if ( ( (unsigned int) (unsigned char) node != 36) && ((unsigned int) (unsigned char)node != 129) &&
    //     ( (unsigned int) (unsigned char) node != 46)  && ( (unsigned int) (unsigned char) node != 116))
    if( lenght <= 1.0 )
    {
        Qi.x = P.x / 2.0 + ( P.x + P_next.x ) / 4.0;

        Qi.y = P.y / 2.0 + ( P.y + P_next.y ) / 4.0;
    }
    else
    {
        Qi.x = ( 7.0 * P.x ) / 8.0 + ( P_next.x ) / 8.0;

        Qi.y = ( 7.0 * P.y ) / 8.0 + ( P_next.y ) / 8.0;
    }

    return Qi;
}


template< typename T >
__device__ T getR_i( int i, T* diagram, int edge_count, char node )
{
    T P;
    int p_index = i % edge_count;
    if( p_index < 0 )
    {
        p_index = p_index + edge_count;
    }
    P.x = diagram[ p_index ].x;
    P.y = diagram[ p_index ].y;

    T P_next;
    P_next.x = diagram[ ( i + 1 ) % edge_count ].x;
    P_next.y = diagram[ ( i + 1 ) % edge_count ].y;

    T Ri;

    float lenght = sqrt( ( ( P_next.x - P.x ) * ( P_next.x - P.x ) + ( P_next.y - P.y ) * ( P_next.y - P.y ) ) );

    //if ( ( (unsigned int) (unsigned char) node != 36) && ((unsigned int) (unsigned char)node != 129) &&
    //     ( (unsigned int) (unsigned char) node != 46)  && ( (unsigned int) (unsigned char) node != 116))
    if( lenght <= 1.0 )
    {
        Ri.x = P_next.x / 2.0 + ( P.x + P_next.x ) / 4.0;

        Ri.y = P_next.y / 2.0 + ( P.y + P_next.y ) / 4.0;
    }
    else
    {
        Ri.x = ( 1.0 * P.x ) / 8.0 + ( 7.0 * P_next.x ) / 8.0;

        Ri.y = ( 1.0 * P.y ) / 8.0 + ( 7.0 * P_next.y ) / 8.0;
    }

    return Ri;
}


template< typename T >
__device__ T getR_i_from_linked_cell( T* diagram, T p, int link_index, int* edge_count_v, int edge_count,
                                      int node_index, int width, char node )
{
    int linked_cell_index = c_neighbor_index( node_index, link_index, width );
    T* linked_cell = &diagram[ linked_cell_index * CELL_SIZE ];
    int linked_edge_count = edge_count_v[ linked_cell_index ];

    int op_index = getPointIndex( getOppositePoint( p, link_index ), linked_cell, linked_edge_count );

    /* Get Ri point for the previous edge */
    return getOppositePointCoord( getR_i( op_index - 1, linked_cell, linked_edge_count, node ), link_index );
    //return getR_i( op_index - 1, linked_cell, linked_cell.size());
}


template< typename T >
__device__ T getQ_i_from_linked_cell( T* diagram, T p, int link_index, int* edge_count_v, int edge_count,
                                      int node_index, int width, char node )
{
    int linked_cell_index = c_neighbor_index( node_index, link_index, width );
    T* linked_cell = &diagram[ linked_cell_index * CELL_SIZE ];
    int linked_edge_count = edge_count_v[ linked_cell_index ];

    int op_index = getPointIndex( getOppositePoint( p, link_index ), linked_cell, linked_edge_count );

    /* Get Qi point for the current edge */
    return getOppositePointCoord( getQ_i( op_index, linked_cell, linked_edge_count, node ), link_index );
    //return getQ_i( op_index, linked_cell, linked_cell.size());
}


__device__ bool isColorEqual( char* c1, char* c2 )
{
    if( ( c1[ 0 ] == c2[ 0 ] ) && ( c1[ 1 ] == c2[ 1 ] ) && ( c1[ 2 ] == c2[ 2 ] ) )
    {
        return true;
    }
    else
    {
        return false;
    }
}


template< typename T >
__device__ bool checkTJunction( char* image_data, int width, int img_widthstep, int height, int i, int j, T p )
{
    /*     Colors
      +----+----+----+
      |    |    |    |
      | c0 | c1 | c2 |
      +----+----+----+
      |    |    |    |
      | c3 | c  | c4 |
      +----+----+----+
      |    |    |    |
      | c5 | c6 | c7 |
      +----+----+----+   */

    int index = j * img_widthstep + i * 3;

    if( ( ( index - img_widthstep - 1 ) < 0 ) || ( ( index + width + 1 ) > ( ( height * img_widthstep ) - 1 ) ) )
    {
        return true;
    }

    char* c0, * c1, * c2, * c3, * c4, * c5, * c6, * c7;

    //c = &image_data[index];
    c0 = &image_data[ index + img_widthstep - 3 ];
    c1 = &image_data[ index + img_widthstep ];
    c2 = &image_data[ index + img_widthstep + 3 ];
    c3 = &image_data[ index - 3 ];
    c4 = &image_data[ index + 3 ];
    c5 = &image_data[ index - img_widthstep - 3 ];
    c6 = &image_data[ index - img_widthstep ];
    c7 = &image_data[ index - img_widthstep + 3 ];


    if( ( p.x == 0.0 ) && ( p.y == 0.0 ) )
    {
        if( !isColorEqual( c3, c5 ) ||
            !isColorEqual( c5, c6 ) )
        {
            return true;
        }
    }

    if( ( p.x == 1.0 ) && ( p.y == 0.0 ) )
    {
        if( !isColorEqual( c4, c7 ) ||
            !isColorEqual( c7, c6 ) )
        {
            return true;
        }
    }

    if( ( p.x == 1.0 ) && ( p.y == 1.0 ) )
    {
        if( !isColorEqual( c1, c2 ) ||
            !isColorEqual( c2, c4 ) )
        {
            return true;
        }
    }

    if( ( p.x == 0.0 ) && ( p.y == 1.0 ) )
    {
        if( !isColorEqual( c0, c1 ) ||
            !isColorEqual( c1, c3 ) )
        {
            return true;
        }
    }

    return false;
}


template< typename T >
__device__ bool isLinkedEdge( T* cell, char node, int index, int edge_count, int i_, int j_, int* link_index,
                              int& li_index )
{
    /* Try the eight possible links of the node */

    /* Edge Slope */

    float slope = ( cell[ mod( index + 1, edge_count ) ].y - cell[ index ].y ) /
                  ( cell[ mod( index + 1, edge_count ) ].x - cell[ index ].x );

    float mid_y = ( cell[ index ].y + cell[ mod( index + 1, edge_count ) ].y ) / 2.0;

    /*      UP
      +----+----+----+
      |    | |  |    |
      | 0  | |  | 2  |
      +----+----+----+
      |    | |  |    |
      | 3  | |  | 4  |
      +----+----+----+
      |    |    |    |
      | 5  | 6  | 7  |
      +----+----+----+   */

    if( ( cell[ index ].y == cell[ mod( index + 1, edge_count ) ].y ) &&
        ( cell[ index ].y > 0.5 ) &&
        ( CHECK_BIT( node, 1 ) ) )
    {
        link_index[ li_index++ ] = 1;
        return true;
    }


    /*      Right
      +----+----+----+
      |    |    |    |
      | 0  | 1  | 2  |
      +----+----+----+
      |    |    |    |
      | 3  |  --|----|
      +----+----+----+
      |    |    |    |
      | 5  | 6  | 7  |
      +----+----+----+   */

    if( ( cell[ index ].x == cell[ mod( index + 1, edge_count ) ].x ) &&
        ( cell[ index ].x > 0.5 ) &&
        ( CHECK_BIT( node, 4 ) ) )
    {
        link_index[ li_index++ ] = 4;
        return true;
    }

    /*      Down
      +----+----+----+
      |    |    |    |
      | 0  | 1  | 2  |
      +----+----+----+
      |    | |  |    |
      | 3  | |  | 4  |
      +----+----+----+
      |    | |  |    |
      | 5  | |  | 7  |
      +----+----+----+   */

    if( ( cell[ index ].y == cell[ mod( index + 1, edge_count ) ].y ) &&
        ( cell[ index ].y < 0.5 ) &&
        ( CHECK_BIT( node, 6 ) ) )
    {
        link_index[ li_index++ ] = 6;
        return true;
    }

    /*      Left
      +----+----+----+
      |    |    |    |
      | 0  | 1  | 2  |
      +----+----+----+
      |    |    |    |
      |----|--  | 4  |
      +----+----+----+
      |    |    |    |
      | 5  | 6  | 7  |
      +----+----+----+   */

    if( ( cell[ index ].x == cell[ mod( index + 1, edge_count ) ].x ) &&
        ( cell[ index ].x < 0.5 ) &&
        ( CHECK_BIT( node, 3 ) ) )
    {
        link_index[ li_index++ ] = 3;
        return true;
    }

    /*    Up-Left
      +----+----+----+
      | \  |    |    |
      |  \ | 1  | 2  |
      +----+----+----+
      |    |\   |    |
      | 3  |    | 4  |
      +----+----+----+
      |    |    |    |
      | 5  | 6  | 7  |
      +----+----+----+   */

    if( ( slope == 1 ) &&
        ( mid_y > 0.5 ) &&
        ( CHECK_BIT( node, 0 ) ) )
    {
        link_index[ li_index++ ] = 0;
        return true;
    }


    /*    Up-Right
      +----+----+----+
      |    |    | /  |
      | 0  | 1  |/   |
      +----+----+----+
      |    |  / |    |
      | 3  |    | 4  |
      +----+----+----+
      |    |    |    |
      | 5  | 6  | 7  |
      +----+----+----+   */

    if( ( slope == -1 ) &&
        ( mid_y > 0.5 ) &&
        ( CHECK_BIT( node, 2 ) ) )
    {
        link_index[ li_index++ ] = 2;
        return true;
    }


    /*    Down-Right
      +----+----+----+
      |    |    |    |
      | 0  | 1  | 2  |
      +----+----+----+
      |    |    |    |
      | 3  |  \ | 4  |
      +----+----+----+
      |    |    |\   |
      | 5  | 6  | \  |
      +----+----+----+   */

    if( ( slope == 1 ) &&
        ( mid_y < 0.5 ) &&
        ( CHECK_BIT( node, 7 ) ) )
    {
        link_index[ li_index++ ] = 7;
        return true;
    }


    /*    Down-Right
      +----+----+----+
      |    |    |    |
      | 0  | 1  | 2  |
      +----+----+----+
      |    |    |    |
      | 3  |/   | 4  |
      +----+----+----+
      |   /|    |    |
      |  / | 6  | 7  |
      +----+----+----+   */

    if( ( slope == -1 ) &&
        ( mid_y < 0.5 ) &&
        ( CHECK_BIT( node, 5 ) ) )
    {
        link_index[ li_index++ ] = 5;
        return true;
    }

    link_index[ li_index++ ] = -1;
    return false;
}


template< typename T >
__device__ T getOppositePoint( T p, int edge )
{
    T result;
    switch( edge )
    {
        case 0:
            result.x = p.x + 1;
            result.y = p.y - 1;
            break;

        case 1:
            result.x = p.x;
            result.y = p.y - 1;
            break;

        case 2:
            result.x = p.x - 1;
            result.y = p.y - 1;
            break;

        case 3:
            result.x = p.x + 1;
            result.y = p.y;
            break;

        case 4:
            result.x = p.x - 1;
            result.y = p.y;
            break;

        case 5:
            result.x = p.x + 1;
            result.y = p.y + 1;
            break;

        case 6:
            result.x = p.x;
            result.y = p.y + 1;
            break;

        case 7:
            result.x = p.x - 1;
            result.y = p.y + 1;
            break;
    }
    return result;
}


template< typename T >
__device__ T getOppositePointCoord( T p, int edge )
{
    T result;
    switch( edge )
    {
        case 0:
            result.x = p.x - 1;
            result.y = p.y + 1;
            break;

        case 1:
            result.x = p.x;
            result.y = p.y + 1;
            break;

        case 2:
            result.x = p.x + 1;
            result.y = p.y + 1;
            break;

        case 3:
            result.x = p.x - 1;
            result.y = p.y;
            break;

        case 4:
            result.x = p.x + 1;
            result.y = p.y;
            break;

        case 5:
            result.x = p.x - 1;
            result.y = p.y - 1;
            break;

        case 6:
            result.x = p.x;
            result.y = p.y - 1;
            break;

        case 7:
            result.x = p.x + 1;
            result.y = p.y - 1;
            break;
    }
    return result;
}


template< typename T >
__device__ int getPointIndex( T p, T* cell, int edge_count )
{
    for( int i = 0; i < edge_count; i++ )
    {
        if( ( p.x == cell[ i ].x ) && ( p.y == cell[ i ].y ) )
        {
            return i;
        }
    }
    return 0;
}


template< typename T >
__device__ void checkEdges( bool* edge_status, T* cell, char node, int edge_count, int i_, int j_, int* link_index )
{
    int li_index = 0;

    /* Loop through each edge */
    for( int i = 0; i < edge_count; i++ )
    {
        edge_status[ i ] = isLinkedEdge( cell, node, i, edge_count, i_, j_, link_index, li_index );
    }
}


template< typename T >
__device__ T midPoint( T p1, T p2 )
{
    T result;
    result.x = ( ( p1.x + p2.x ) / 2.0 );
    result.y = ( ( p1.y + p2.y ) / 2.0 );
    return result;
}


template< typename T >
__device__ int subdivision( char* image_data, T* cell, T* diagram, int* edge_count_v, int node_index,
                            int width, int img_widthstep, int height, int i_, int j_, char node, bool* edge_status,
                            int* link_index )
{
    //T d_copy[CELL_SIZE];
//    bool edge_status[CELL_SIZE/2];
//    int link_index[CELL_SIZE/2];

    int edge_index = 0;
    int edge_count = edge_count_v[ node_index ];

    checkEdges( edge_status, &diagram[ ( node_index * CELL_SIZE ) ], node, edge_count, i_, j_, link_index );

    T Qi, R_previous;

    for( int i = 0; i < edge_count; i++ )
    {
        /* Two adjacent edges that are border */
        if( !edge_status[ i ] && !edge_status[ mod( i - 1, edge_count ) ] )
        {
            if( checkTJunction( image_data, width, img_widthstep, height, i_, j_,
                                diagram[ ( node_index * CELL_SIZE ) + i ] ) )
            {
                cell[ edge_index ].x = diagram[ ( node_index * CELL_SIZE ) + i ].x;
                cell[ edge_index++ ].y = diagram[ ( node_index * CELL_SIZE ) + i ].y;
            }
            else
            {
                Qi = getQ_i( i, &diagram[ ( node_index * CELL_SIZE ) ], edge_count, node );

                R_previous = getR_i( i - 1, &diagram[ ( node_index * CELL_SIZE ) ], edge_count, node );

                cell[ edge_index ].x = R_previous.x;
                cell[ edge_index++ ].y = R_previous.y;

                cell[ edge_index ].x = Qi.x;
                cell[ edge_index++ ].y = Qi.y;
            }
        }
        else
        /* Current edge is internal and the following is border
           In this case we have to take the Ri from the previous edge of the adjacent cell */
        if( !edge_status[ i ] && edge_status[ mod( i - 1, edge_count ) ] )
        {
            Qi = getQ_i( i, &diagram[ ( node_index * CELL_SIZE ) ], edge_count, node );

            T R_adjacent =
                getR_i_from_linked_cell( diagram, diagram[ ( node_index * CELL_SIZE ) + i ],
                                         link_index[ mod( i - 1,
                                                          edge_count ) ], edge_count_v, edge_count, node_index, width,
                                         node );

            R_adjacent = midPoint( Qi, R_adjacent );

            cell[ edge_index ].x = R_adjacent.x;
            cell[ edge_index++ ].y = R_adjacent.y;

            cell[ edge_index ].x = Qi.x;
            cell[ edge_index++ ].y = Qi.y;
        }
        else
        /* Current edge is border and the following is internal
           In this case we have to take the Qi from the next edge of the adjacent cell */
        if( edge_status[ i ] && !edge_status[ mod( i - 1, edge_count ) ] )
        {
            T Q_adjacent = getQ_i_from_linked_cell( diagram, diagram[ ( node_index * CELL_SIZE ) + i ], link_index[ i ],
                                                    edge_count_v, edge_count, node_index, width, node );

//                    if ( (i_ == 3) && (j_ == 5) )
//                        printf("Q_adjacent for i=%d, j=%d = P( %2.2f, %2.2f )\n", i_, j_, Q_adjacent.x, Q_adjacent.y);

            T Ri = getR_i( i - 1, &diagram[ ( node_index * CELL_SIZE ) ], edge_count, node );

            Q_adjacent = midPoint( Ri, Q_adjacent );

            cell[ edge_index ].x = Ri.x;
            cell[ edge_index++ ].y = Ri.y;

            cell[ edge_index ].x = Q_adjacent.x;
            cell[ edge_index++ ].y = Q_adjacent.y;
        }
        else
        /* Two adjacent internal edges. Nothing to do */
        {
            cell[ edge_index ].x = diagram[ ( node_index * CELL_SIZE ) + i ].x;
            cell[ edge_index++ ].y = diagram[ ( node_index * CELL_SIZE ) + i ].y;
        }
    }

//    if ( (i_ == 3) && (j_ == 5) )
//    {
//        for(int i = 0; i < (edge_index); i++)
//        {
//            printf(" subdividion  P( %2.2f, %2.2f )\n", cell[i].x, cell[i].y);
//        }
//    }

    //free(d_copy);
    //free(edge_status);
    //free(link_index);

    return edge_index;
    //return edge_count_v[node_index];
    //return edge_count * 2;
}


